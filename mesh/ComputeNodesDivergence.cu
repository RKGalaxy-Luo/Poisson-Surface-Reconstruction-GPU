#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   ComputeNodesDivergence.cu
 * \brief  ����ڵ�ɢ��cuda����ʵ��
 * 
 * \author LUOJIAXUAN
 * \date   May 24th 2024
 *********************************************************************/
#include "ComputeNodesDivergence.h"
#if defined(__HIPCC__)		//�����NVCC����������
#include <hipcub/hipcub.hpp>
#endif

namespace SparseSurfelFusion {
	namespace device {
		__device__ __constant__ int maxDepth = MAX_DEPTH_OCTREE;	// Octree������

		__device__ __constant__ int res = RESOLUTION;				// �ֱ���

		__device__ __constant__ int decodeOffset_1 = (1 << (MAX_DEPTH_OCTREE + 1));

		__device__ __constant__ int decodeOffset_2 = (1 << (2 * (MAX_DEPTH_OCTREE + 1)));
	}
}

__global__ void SparseSurfelFusion::device::computeFinerNodesDivergenceKernel(DeviceArrayView<int> BaseAddressArray, DeviceArrayView<int> encodeNodeIndexInFunction, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<Point3D<float>> VectorField, DeviceArrayView<double> dot_F_DF, const unsigned int begin, const unsigned int calculatedNodeNum, float* Divergence)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= calculatedNodeNum)	return;
	const unsigned int offset = begin + idx;
	const unsigned int startDLevel = BaseAddressArray[device::maxDepth];
	double val = 0;
#pragma unroll
	for (int i = 0; i < 27; i++) {
		int neighborIdx = NodeArray[offset].neighs[i];
		if (neighborIdx == -1)	continue;
		for (int j = 0; j < NodeArray[neighborIdx].dnum; j++) {						// ������ǰ�ڵ���ھӽڵ���maxDepth���������ĵ�����Ҷ�ӽڵ�
			int NodeIndexDLevel = NodeArray[neighborIdx].didx + j;					// ��maxDepth��Ҷ�ӽڵ��index
			const Point3D<float>& vo = VectorField[NodeIndexDLevel];
			int idxO_1[3], idxO_2[3];

			int encodeIndex = encodeNodeIndexInFunction[offset];						// ��õ�ǰ�ڵ����������
			idxO_1[0] = encodeIndex % decodeOffset_1;								// ȡ�������11λ	[0 , 10]
			idxO_1[1] = (encodeIndex / decodeOffset_1) % decodeOffset_1;			// ȡ�����м�11λ	[11, 21]
			idxO_1[2] = encodeIndex / decodeOffset_2;								// ȡ������ǰ10λ	[22, 31]

			encodeIndex = encodeNodeIndexInFunction[startDLevel + NodeIndexDLevel];	// ��õ�ǰ�ڵ���maxdepth���Ҷ�ӽڵ����������
			idxO_2[0] = encodeIndex % decodeOffset_1;								// ȡ�������11λ	[0 , 10]
			idxO_2[1] = (encodeIndex / decodeOffset_1) % decodeOffset_1;			// ȡ�����м�11λ	[11, 21]
			idxO_2[2] = encodeIndex / decodeOffset_2;								// ȡ������ǰ10λ	[22, 31]

			int scratch[3];
			scratch[0] = idxO_1[0] + idxO_2[0] * res;
			scratch[1] = idxO_1[1] + idxO_2[1] * res;
			scratch[2] = idxO_1[2] + idxO_2[2] * res;

			Point3D<float> uo;
			uo.coords[0] = dot_F_DF[scratch[0]];
			uo.coords[1] = dot_F_DF[scratch[1]];
			uo.coords[2] = dot_F_DF[scratch[2]];

			val += DotProduct(vo, uo);
		}
	}
	Divergence[offset] = val;	// ��ǰ�ڵ�ɢ��
}

__device__ float SparseSurfelFusion::device::DotProduct(const Point3D<float>& p1, const Point3D<float>& p2)
{
	float ans = 0;
	ans += p1.coords[0] * p2.coords[0];
	ans += p1.coords[1] * p2.coords[1];
	ans += p1.coords[2] * p2.coords[2];
	return ans;
}

__global__ void SparseSurfelFusion::device::computeCoverNums(DeviceArrayView<OctNode> NodeArray, const unsigned int index, unsigned int* coverNums)
{
	coverNums[0] = 0;
	for (int i = 0; i < 27; i++) {
		int neighbor = NodeArray[index].neighs[i];	// �ھӽڵ�idx
		if (neighbor != -1) {
			coverNums[i + 1] = NodeArray[neighbor].dnum + coverNums[i];
		}
		else {
			coverNums[i + 1] = coverNums[i];
		}
	}
}

__global__ void SparseSurfelFusion::device::generateDLevelIndexArrayKernel(DeviceArrayView<OctNode> NodeArray, const unsigned int index, const unsigned int* coverNums, const unsigned int totalCoverNum, unsigned int* DLevelIndexArray)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= totalCoverNum)	return;

	int neighborIdx;	// �жϵ�ǰ���D��ڵ���������һ���ϲ�ڵ�(1-4����ڵ㼰���ھӵ�27����)
	for (neighborIdx = 0; neighborIdx < 27; neighborIdx++) {
		if (coverNums[neighborIdx] <= idx && idx < coverNums[neighborIdx + 1]) {
			break;
		}
	}
	int Current27NodesDLevelStartIndex = NodeArray[NodeArray[index].neighs[neighborIdx]].didx;	// ��ǰ���neighborIdx�ڵ���D���е���ʼλ��
	DLevelIndexArray[idx] = Current27NodesDLevelStartIndex + idx - coverNums[neighborIdx];		// idx - coverNums[neighborIdx]��������ڵ�ǰneighborIdx�ڵ���ʵλ�õľ���
}

__global__ void SparseSurfelFusion::device::computeCoarserNodesDivergenceKernel(DeviceArrayView<int> BaseAddressArrayDevice, DeviceArrayView<int> encodeNodeIndexInFunction, DeviceArrayView<Point3D<float>> VectorField, DeviceArrayView<double> dot_F_DF, const unsigned int index, const unsigned int* DLevelIndexArray, const unsigned int totalCoverNum, float* divg)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= totalCoverNum)	return;
	const unsigned int startD = BaseAddressArrayDevice[device::maxDepth];
	int DIdx = DLevelIndexArray[idx];
	const Point3D<float>& vo = VectorField[DIdx];

	int idxO_1[3], idxO_2[3];

	int encodeIdx = encodeNodeIndexInFunction[index];
	idxO_1[0] = encodeIdx % decodeOffset_1;
	idxO_1[1] = (encodeIdx / decodeOffset_1) % decodeOffset_1;
	idxO_1[2] = encodeIdx / decodeOffset_2;

	encodeIdx = encodeNodeIndexInFunction[startD + DIdx];
	idxO_2[0] = encodeIdx % decodeOffset_1;
	idxO_2[1] = (encodeIdx / decodeOffset_1) % decodeOffset_1;
	idxO_2[2] = encodeIdx / decodeOffset_2;

	int scratch[3];
	scratch[0] = idxO_1[0] + idxO_2[0] * device::res;
	scratch[1] = idxO_1[1] + idxO_2[1] * device::res;
	scratch[2] = idxO_1[2] + idxO_2[2] * device::res;

	Point3D<float> uo;
	uo.coords[0] = dot_F_DF[scratch[0]];
	uo.coords[1] = dot_F_DF[scratch[1]];
	uo.coords[2] = dot_F_DF[scratch[2]];

	divg[idx] = DotProduct(vo, uo);
}

void SparseSurfelFusion::ComputeNodesDivergence::computeFinerNodesDivergence(DeviceArrayView<int> BaseAddressArrayDevice, DeviceArrayView<int> encodeNodeIndexInFunction, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<Point3D<float>> VectorField, DeviceArrayView<double> dot_F_DF, const unsigned int left, const unsigned int right, hipStream_t stream)
{
//#ifdef CHECK_MESH_BUILD_TIME_COST
//	auto start = std::chrono::high_resolution_clock::now();						// ��¼��ʼʱ���
//#endif // CHECK_MESH_BUILD_TIME_COST

	Divergence.ResizeArrayOrException(NodeArray.Size());	// ��NodeArray��Сһ��
	const unsigned int CalculatedNodeNum = right - left;	// �������Ľڵ�����
	dim3 block(128);
	dim3 grid(divUp(CalculatedNodeNum, block.x));
	device::computeFinerNodesDivergenceKernel << <grid, block, 0, stream >> > (BaseAddressArrayDevice, encodeNodeIndexInFunction, NodeArray, VectorField, dot_F_DF, left, CalculatedNodeNum, Divergence.Array().ptr());

//#ifdef CHECK_MESH_BUILD_TIME_COST
//	CHECKCUDA(hipStreamSynchronize(stream));
//	auto end = std::chrono::high_resolution_clock::now();							// ��¼����ʱ���
//	std::chrono::duration<double, std::milli> duration = end - start;				// ����ִ��ʱ�䣨��msΪ��λ��
//	std::cout << "����[CoarserLevelNum + 1, maxDepth]��ڵ�ɢ�ȵ�ʱ��: " << duration.count() << " ms" << std::endl;	// ���
//#endif // CHECK_MESH_BUILD_TIME_COST

}

void SparseSurfelFusion::ComputeNodesDivergence::computeCoarserNodesDivergence(const int* BaseAddressArray, DeviceArrayView<int> BaseAddressArrayDevice, DeviceArrayView<int> encodeNodeIndexInFunction, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<Point3D<float>> VectorField, DeviceArrayView<double> dot_F_DF, const unsigned int left, const unsigned int right, hipStream_t stream)
{
//#ifdef CHECK_MESH_BUILD_TIME_COST
//	auto start = std::chrono::high_resolution_clock::now();						// ��¼��ʼʱ���
//#endif // CHECK_MESH_BUILD_TIME_COST

	Divergence.ResizeArrayOrException(NodeArray.Size());	// ��NodeArray��Сһ��
	const unsigned int CalculatedNodeNum = right - left;	// �������Ľڵ�����
	for (int i = left; i < CalculatedNodeNum; i++) {			// [0, CoarserLevelNum]��ڵ�����
		int depth = 0;	// ��ǰ�ڵ����
		for (int j = 0; j <= COARSER_DIVERGENCE_LEVEL_NUM; j++) {
			if (BaseAddressArray[j] <= i && i < BaseAddressArray[j + 1]) {
				depth = j;
				break;
			}
		}
		unsigned int* coverNums = NULL;
		CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&coverNums), sizeof(unsigned int) * 28, stream));

		device::computeCoverNums << <1, 1, 0, stream >> > (NodeArray, i, coverNums);
		unsigned int totalCoverNum;		// ��ǰ�ڵ㼰���ھӸ��ǵ�maxDepth��Ľڵ�����
		CHECKCUDA(hipMemcpyAsync(&totalCoverNum, coverNums + 27, sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
		
		float* divg = NULL;
		CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&divg), sizeof(float) * totalCoverNum, stream));

		unsigned int* DLevelIndexArray = NULL;	// ����idx = [0, totalCoverNum)��D��ڵ㣬����Щ�ڵ�ӳ�䵽D���Ӧ��D_idx�ϡ����磺idx = 3��D��ڵ���NodeArray�е�λ�ÿ���Ϊ100��
		CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&DLevelIndexArray), sizeof(unsigned int) * totalCoverNum, stream));

		dim3 block(128);
		dim3 grid(divUp(totalCoverNum, block.x));
		device::generateDLevelIndexArrayKernel << <grid, block, 0, stream >> > (NodeArray, i, coverNums, totalCoverNum, DLevelIndexArray);
		device::computeCoarserNodesDivergenceKernel << <grid, block, 0, stream >> > (BaseAddressArrayDevice, encodeNodeIndexInFunction, VectorField, dot_F_DF, i, DLevelIndexArray, totalCoverNum, divg);
		// ��Լ�ӷ�
		float* divgSum = NULL;
		CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&divgSum), sizeof(float), stream));

		void* d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, divg, divgSum, totalCoverNum, stream);
		CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
		hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, divg, divgSum, totalCoverNum, stream);
		CHECKCUDA(hipMemcpyAsync(Divergence.Array().ptr() + i, divgSum, sizeof(float), hipMemcpyDeviceToDevice, stream));
		CHECKCUDA(hipFreeAsync(coverNums, stream));			// ��ʱ����, ���꼴ɾ
		CHECKCUDA(hipFreeAsync(divg, stream));					// ��ʱ����, ���꼴ɾ
		CHECKCUDA(hipFreeAsync(DLevelIndexArray, stream));		// ��ʱ����, ���꼴ɾ
		CHECKCUDA(hipFreeAsync(divgSum, stream));				// ��ʱ����, ���꼴ɾ
		CHECKCUDA(hipFreeAsync(d_temp_storage, stream));		// ��ʱ����, ���꼴ɾ
	}

//#ifdef CHECK_MESH_BUILD_TIME_COST
//	// ���в��������ͬ��
//	for (int i = 0; i < MAX_MESH_STREAM - 1; i++) {
//		CHECKCUDA(hipStreamSynchronize(streams[i]));
//	}
//	auto end = std::chrono::high_resolution_clock::now();							// ��¼����ʱ���
//	std::chrono::duration<double, std::milli> duration = end - start;				// ����ִ��ʱ�䣨��msΪ��λ��
//	std::cout << "����[1, CoarserLevelNum]��ڵ�ɢ�ȵ�ʱ��: " << duration.count() << " ms" << std::endl;	// ���
//#endif // CHECK_MESH_BUILD_TIME_COST
}
