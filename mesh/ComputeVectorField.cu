#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   ComputeVectorField.cu
 * \brief  ����������cuda����ʵ��
 * 
 * \author LUOJIAXUAN
 * \date   May 15th 2024
 *********************************************************************/
#include "ComputeVectorField.h"

namespace SparseSurfelFusion {
	namespace device {
		__device__ __constant__ int maxDepth = MAX_DEPTH_OCTREE;

		__device__ __constant__ int normalize = NORMALIZE;
	}
}

__device__ float SparseSurfelFusion::device::FCenterWidthPoint(int idx, int i, int j, const ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2>& BaseFunctionMaxDepth_d, const Point3D<float>& center, const float& width, const Point3D<float>& point)
{
	ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2> thisFunction_X = BaseFunctionMaxDepth_d.shift(center.coords[0]);
	ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2> thisFunction_Y = BaseFunctionMaxDepth_d.shift(center.coords[1]);
	ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2> thisFunction_Z = BaseFunctionMaxDepth_d.shift(center.coords[2]);
	float x = value(thisFunction_X, point.coords[0]);
	float y = value(thisFunction_Y, point.coords[1]);
	float z = value(thisFunction_Z, point.coords[2]);
	float ret = x * y * z;
	switch (device::normalize) {
	case 2:
		ret /= sqrt(1.0 / (1 << (device::maxDepth)));
		break;
	case 1:
		ret /= 1.0 / (1 << (device::maxDepth));
		break;
	}
	//if (/*1000 <= idx && idx < 1100*/idx == 1050) {
	//	printf("index = %d  (%d, %d)  point = (%.10f, %.10f, %.10f)  o_c = (%.5f, %.5f, %.5f)  ret = %.10f\n", idx, i, j, point.coords[0], point.coords[1], point.coords[2], center.coords[0], center.coords[1], center.coords[2], ret);
	//}
	return ret;
}

__device__ void SparseSurfelFusion::device::getFunctionIdxNode(const int& key, const int& maxDepth, int* index)
{
	// (����device::maxDepth = 8)
	index[0] = (1 << device::maxDepth) - 1;	// ��ֵ:00011111111 
	index[1] = index[0];
	index[2] = index[1];

	// (1 << (device::maxDepth - depth)) = 00011111111

	for (int depth = device::maxDepth; depth >= 1; depth--) {
		// ��ñ����x,y,z�ķ���
		int sonKeyX = (key >> (3 * (device::maxDepth - depth) + 2)) & 1;  // ��ú��ӽڵ�Key��X����
		int sonKeyY = (key >> (3 * (device::maxDepth - depth) + 1)) & 1;  // ��ú��ӽڵ�Key��Y����
		int sonKeyZ = (key >> (3 * (device::maxDepth - depth))) & 1;	  // ��ú��ӽڵ�Key��Z����
		index[0] += sonKeyX * (1 << (device::maxDepth - depth));
		index[1] += sonKeyY * (1 << (device::maxDepth - depth));
		index[2] += sonKeyZ * (1 << (device::maxDepth - depth));
	}
}

__global__ void SparseSurfelFusion::device::CalculateVectorFieldKernel(ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2>* BaseFunctionMaxDepth_Device, DeviceArrayView<OrientedPoint3D<float>> DenseOrientedPoints, DeviceArrayView<OctNode> NodeArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeNum, Point3D<float>* VectorField)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= DLevelNodeNum) return;
	const unsigned int offset = DLevelOffset + idx;
	int index[3];
	float width;
	getFunctionIdxNode(NodeArray[offset].key, device::maxDepth, index);
	Point3D<float> o_c;
	BinaryNode<float>::CenterAndWidth(index[0], o_c.coords[0], width);
	BinaryNode<float>::CenterAndWidth(index[1], o_c.coords[1], width);
	BinaryNode<float>::CenterAndWidth(index[2], o_c.coords[2], width);

	//if (5000 <= idx && idx < 5100) {
	//	printf("index = %d   width = %.7f   o_c(%.7f, %.7f, %.7f)\n", idx, width, o_c.coords[0], o_c.coords[1], o_c.coords[2]);
	//}

	/** ���index�Լ�o_c,���� **/

	Point3D<float> val;
	for (int i = 0; i < 27; i++) {
		int neighbor = NodeArray[offset].neighs[i];
		if (neighbor != -1) {
			for (int j = 0; j < NodeArray[neighbor].pnum; j++) {
				int pointIdx = NodeArray[neighbor].pidx + j;	// �ڳ��ܵ������е�λ��
				float weight = FCenterWidthPoint(idx, i, j, *BaseFunctionMaxDepth_Device, o_c, width, DenseOrientedPoints[pointIdx].point);
				val.coords[0] += weight * DenseOrientedPoints[pointIdx].normal.coords[0];
				val.coords[1] += weight * DenseOrientedPoints[pointIdx].normal.coords[1];
				val.coords[2] += weight * DenseOrientedPoints[pointIdx].normal.coords[2];
				//if (idx == 995) {
				//	printf("(%d, %d)  pnum = %d  pidx = %d  pointIdx[%d] = (%.10f, %.10f, %.10f, %.10f, %.10f, %.10f)\n", i, j, NodeArray[neighbor].pnum, NodeArray[neighbor].pidx, pointIdx, DenseOrientedPoints[pointIdx].point.coords[0], DenseOrientedPoints[pointIdx].point.coords[1], DenseOrientedPoints[pointIdx].point.coords[2], DenseOrientedPoints[pointIdx].normal.coords[0], DenseOrientedPoints[pointIdx].normal.coords[1], DenseOrientedPoints[pointIdx].normal.coords[2]);
				//}
			}
		}
	}

	VectorField[idx].coords[0] += val.coords[0];
	VectorField[idx].coords[1] += val.coords[1];
	VectorField[idx].coords[2] += val.coords[2];
}

void SparseSurfelFusion::ComputeVectorField::CalculateVectorField(ConfirmedPPolynomial<CONVTIMES, CONVTIMES + 2>* BaseFunctionMaxDepth_Device, DeviceArrayView<OrientedPoint3D<float>> DenseOrientedPoints, DeviceArrayView<OctNode> NodeArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeNum, DeviceBufferArray<Point3D<float>>& VectorField, hipStream_t stream)
{
	dim3 block(128);
	dim3 grid(divUp(DLevelNodeNum, block.x));
	device::CalculateVectorFieldKernel << <grid, block, 0, stream >> > (BaseFunctionMaxDepth_Device, DenseOrientedPoints, NodeArray, DLevelOffset, DLevelNodeNum, VectorField.Array().ptr());
}