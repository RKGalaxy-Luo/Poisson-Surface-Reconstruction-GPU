#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   DrawMesh.cu
 * \brief  OpenGL������Ⱦ����
 * 
 * \author LUOJIAXUAN
 * \date   June 5th 2024
 *********************************************************************/
#include "DrawMesh.h"


__device__ float3 SparseSurfelFusion::device::VectorNormalize(const float3& normal)
{
	float3 result;
	float length = sqrtf(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);
	if (length == 0) { result.x = 0; result.y = 0; result.z = 0; }
	else {
		result.x = normal.x / length;
		result.y = normal.y / length;
		result.z = normal.z / length;
	}
	return result;
}

__device__ float3 SparseSurfelFusion::device::CrossProduct(const float3& Vector_OA, const float3& Vector_OB)
{
	float3 result;
	result.x = Vector_OA.y * Vector_OB.z - Vector_OA.z * Vector_OB.y;
	result.y = Vector_OA.z * Vector_OB.x - Vector_OA.x * Vector_OB.z;
	result.z = Vector_OA.x * Vector_OB.y - Vector_OA.y * Vector_OB.x;
	return result;
}

__global__ void SparseSurfelFusion::device::CalculateMeshNormalsKernel(const Point3D<float>* verticesArray, const TriangleIndex* indicesArray, const unsigned int meshCount, Point3D<float>* normalsArray)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= meshCount) return;
	Point3D<float> Point_O = verticesArray[indicesArray[idx].idx[0]];
	Point3D<float> Point_A = verticesArray[indicesArray[idx].idx[1]];
	Point3D<float> Point_B = verticesArray[indicesArray[idx].idx[2]];
	float3 Vector_OA, Vector_OB;

	Vector_OA.x = Point_A.coords[0] - Point_O.coords[0];
	Vector_OA.y = Point_A.coords[1] - Point_O.coords[1];
	Vector_OA.z = Point_A.coords[2] - Point_O.coords[2];

	Vector_OB.x = Point_B.coords[0] - Point_O.coords[0];
	Vector_OB.y = Point_B.coords[1] - Point_O.coords[1];
	Vector_OB.z = Point_B.coords[2] - Point_O.coords[2];

	float3 Normal = CrossProduct(Vector_OA, Vector_OB);
	float3 normalizedNormal = VectorNormalize(Normal);
	normalsArray[idx].coords[0] = normalizedNormal.x;
	normalsArray[idx].coords[1] = normalizedNormal.y;
	normalsArray[idx].coords[2] = normalizedNormal.z;
}

__global__ void SparseSurfelFusion::device::CountConnectedTriangleNumKernel(const TriangleIndex* indicesArray, const unsigned int meshCount, unsigned int* ConnectedTriangleNum)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= meshCount) return;
	atomicAdd(&ConnectedTriangleNum[indicesArray[idx].idx[0]], 1);
	atomicAdd(&ConnectedTriangleNum[indicesArray[idx].idx[1]], 1);
	atomicAdd(&ConnectedTriangleNum[indicesArray[idx].idx[2]], 1);
}

__global__ void SparseSurfelFusion::device::VerticesNormalsSumKernel(const Point3D<float>* meshNormals, const TriangleIndex* indicesArray, const unsigned int meshCount, Point3D<float>* VerticesNormalsSum)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= meshCount)	return;
	// ����mesh��ÿһ�����㶼Ӧ�ü����䷨��
	// ������Ԫ�ĵ�һ������
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[0]].coords[0], meshNormals[idx].coords[0]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[0]].coords[1], meshNormals[idx].coords[1]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[0]].coords[2], meshNormals[idx].coords[2]);
	// ������Ԫ�ĵڶ�������
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[1]].coords[0], meshNormals[idx].coords[0]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[1]].coords[1], meshNormals[idx].coords[1]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[1]].coords[2], meshNormals[idx].coords[2]);
	// ������Ԫ�ĵ���������
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[2]].coords[0], meshNormals[idx].coords[0]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[2]].coords[1], meshNormals[idx].coords[1]);
	atomicAdd(&VerticesNormalsSum[indicesArray[idx].idx[2]].coords[2], meshNormals[idx].coords[2]);
	//if (idx % 100 == 0) printf("ScaledVerticesNormalsSum[%d] = (%.10f, %.10f, %.10f)\n", idx, meshNormals[idx].coords[0], meshNormals[idx].coords[1], meshNormals[idx].coords[2]);
	//if (idx % 100 == 0) printf("ScaledVerticesNormalsSum[%d] = (%.10f, %.10f, %.10f)\n", idx, ScaledVerticesNormalsSum[idx].coords[0], ScaledVerticesNormalsSum[idx].coords[1], ScaledVerticesNormalsSum[idx].coords[2]);
}

__global__ void SparseSurfelFusion::device::CalculateVerticesAverageNormals(const unsigned int* ConnectedTriangleNum, const Point3D<float>* VerticesNormalsSum, const unsigned int verticesCount, Point3D<float>* VerticesAverageNormals)
{
	const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= verticesCount)	return;
	int meshCount = ConnectedTriangleNum[idx];
	float AverageNormalX = VerticesNormalsSum[idx].coords[0] / meshCount;
	float AverageNormalY = VerticesNormalsSum[idx].coords[1] / meshCount;
	float AverageNormalZ = VerticesNormalsSum[idx].coords[2] / meshCount;
	//if (meshCount == 0) printf("Error = %d\n", idx);
	//if (idx % 1000 == 0) printf("meshCount[%d] = %d\n", idx, meshCount);
	//if (idx % 100 == 0) printf("ScaledVerticesNormalsSum[%d] = (%.10f, %.10f, %.10f)\n", idx, ScaledVerticesNormalsSum[idx].coords[0], ScaledVerticesNormalsSum[idx].coords[1], ScaledVerticesNormalsSum[idx].coords[2]);
	//if (idx % 1000 == 0) printf("AverageNormals[%d] = (%.10f, %.10f, %.10f)\n", idx, AverageNormalX, AverageNormalY, AverageNormalZ);
	
	float3 averageNormal;
	averageNormal.x = AverageNormalX;
	averageNormal.y = AverageNormalY;
	averageNormal.z = AverageNormalZ;

	float3 NormalizedAverageNormal = VectorNormalize(averageNormal);

	VerticesAverageNormals[idx].coords[0] = NormalizedAverageNormal.x;
	VerticesAverageNormals[idx].coords[1] = NormalizedAverageNormal.y;
	VerticesAverageNormals[idx].coords[2] = NormalizedAverageNormal.z;
	//if (idx % 1000 == 0) printf("NormalizedAverageNormals[%d] = (%.10f, %.10f, %.10f)\n", idx, NormalizedAverageNormal.x, NormalizedAverageNormal.y, NormalizedAverageNormal.z);

}

void SparseSurfelFusion::DrawMesh::CalculateMeshNormals(DeviceArrayView<Point3D<float>> meshVertices, DeviceArrayView<TriangleIndex> meshTriangleIndices, hipStream_t stream)
{

#ifdef CHECK_MESH_BUILD_TIME_COST
	auto time1 = std::chrono::high_resolution_clock::now();					// ��¼��ʼʱ���
#endif // CHECK_MESH_BUILD_TIME_COST

	TranglesCount = meshTriangleIndices.Size();
	VerticesCount = meshVertices.Size();
	MeshVertices.ResizeArrayOrException(VerticesCount);
	MeshTriangleIndices.ResizeArrayOrException(TranglesCount);

	CHECKCUDA(hipMemcpyAsync(MeshVertices.Ptr(), meshVertices.RawPtr(), sizeof(Point3D<float>) * VerticesCount, hipMemcpyDeviceToDevice, stream));
	CHECKCUDA(hipMemcpyAsync(MeshTriangleIndices.Ptr(), meshTriangleIndices.RawPtr(), sizeof(TriangleIndex) * TranglesCount, hipMemcpyDeviceToDevice, stream));


	//VerticesNormals.resize(VerticesCount);
	Point3D<float>* MeshNormalsDevice = NULL;	// ��¼����õ�����������ķ���
	CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&MeshNormalsDevice), sizeof(Point3D<float>) * TranglesCount, stream));

	//std::vector<TriangleIndex> TriangleIndexHost;
	//mesh.GetTriangleIndices(TriangleIndexHost);
	//std::cout << "TriangleIndexHostCount = " << TriangleIndexHost.size() << std::endl;

	//TriangleIndex* TriangleIndexDevice = NULL;	// ������������
	//CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&TriangleIndexDevice), sizeof(TriangleIndex) * MeshCount, stream));
	//CHECKCUDA(hipMemcpyAsync(TriangleIndexDevice, TriangleIndexHost.data(), sizeof(Point3D<float>) * MeshCount, hipMemcpyHostToDevice, stream));

	//std::vector<Point3D<float>> VerticesArrayHost;
	//mesh.GetVertexArray(VerticesArrayHost);
	//std::cout << "VerticesArrayHostCount = " << VerticesArrayHost.size() << std::endl;

	//Point3D<float>* VerticesArrayDevice = NULL;		// ��������
	//CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&VerticesArrayDevice), sizeof(Point3D<float>) * VerticesCount, stream));
	//CHECKCUDA(hipMemcpyAsync(VerticesArrayDevice, VerticesArrayHost.data(), sizeof(Point3D<float>) * VerticesCount, hipMemcpyHostToDevice, stream));
	//  << <grid_Mesh, block_Mesh, 0, stream >> > 
	dim3 block_Mesh(256);
	dim3 grid_Mesh(divUp(TranglesCount, block_Mesh.x));
	device::CalculateMeshNormalsKernel << <grid_Mesh, block_Mesh, 0, stream >> > (MeshVertices.ArrayView(), MeshTriangleIndices.ArrayView(), TranglesCount, MeshNormalsDevice);

	unsigned int* ConnectedTriangleNum = NULL;		// ��¼һ�������ж����ڽӵ�������
	CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&ConnectedTriangleNum), sizeof(unsigned int) * VerticesCount, stream));
	CHECKCUDA(hipMemsetAsync(ConnectedTriangleNum, 0, sizeof(unsigned int) * VerticesCount, stream));
	device::CountConnectedTriangleNumKernel << <grid_Mesh, block_Mesh, 0, stream >> > (MeshTriangleIndices.ArrayView(), TranglesCount, ConnectedTriangleNum);

	Point3D<float>* VerticesNormalsSum = NULL;		// ��¼���ڽӵ�����Mesh�ķ���������
	CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&VerticesNormalsSum), sizeof(Point3D<float>) * VerticesCount, stream));
	CHECKCUDA(hipMemsetAsync(VerticesNormalsSum, 0.0f, sizeof(Point3D<float>) * VerticesCount, stream));
	device::VerticesNormalsSumKernel << <grid_Mesh, block_Mesh, 0, stream >> > (MeshNormalsDevice, MeshTriangleIndices.ArrayView(), TranglesCount, VerticesNormalsSum);

	VerticesAverageNormals.ResizeArrayOrException(VerticesCount);

	//// << <grid_vertex, block_vertex, 0, stream >> > 
	//Point3D<float>* VerticesAverageNormals = NULL;	// ��һ���Ķ���ƽ��������
	//CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&VerticesAverageNormals), sizeof(Point3D<float>) * VerticesCount, stream));
	dim3 block_vertex(256);
	dim3 grid_vertex(divUp(VerticesCount, block_vertex.x));
	device::CalculateVerticesAverageNormals << <grid_vertex, block_vertex, 0, stream >> > (ConnectedTriangleNum, VerticesNormalsSum, VerticesCount, VerticesAverageNormals.Ptr());

	//CHECKCUDA(hipMemcpyAsync(VerticesNormals.data(), VerticesAverageNormals, sizeof(Point3D<float>) * VerticesCount, hipMemcpyDeviceToHost, stream));

	CHECKCUDA(hipFreeAsync(MeshNormalsDevice, stream));
	//CHECKCUDA(hipFreeAsync(TriangleIndexDevice, stream));
	//CHECKCUDA(hipFreeAsync(VerticesArrayDevice, stream));
	CHECKCUDA(hipFreeAsync(ConnectedTriangleNum, stream));
	CHECKCUDA(hipFreeAsync(VerticesNormalsSum, stream));

	CHECKCUDA(hipStreamSynchronize(stream));

#ifdef CHECK_MESH_BUILD_TIME_COST
	CHECKCUDA(hipStreamSynchronize(stream));
	auto time2 = std::chrono::high_resolution_clock::now();					// ��¼����ʱ���
	std::chrono::duration<double, std::milli> duration1 = time2 - time1;		// ����ִ��ʱ�䣨��msΪ��λ��
	std::cout << "����Mesh������ʱ��: " << duration1.count() << " ms" << std::endl;
#endif // CHECK_MESH_BUILD_TIME_COST
}
