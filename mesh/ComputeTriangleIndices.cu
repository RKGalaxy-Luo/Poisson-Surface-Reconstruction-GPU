#include "hip/hip_runtime.h"
/*****************************************************************//**
 * \file   ComputeTriangleIndices.cu
 * \brief  �����޸��������񣬹�������
 * 
 * \author LUOJIAXUAN
 * \date   June 3rd 2024
 *********************************************************************/
#include "ComputeTriangleIndices.h"
#if defined(__HIPCC__)		//�����NVCC����������
#include <hipcub/hipcub.hpp>
#endif
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
namespace SparseSurfelFusion {

    struct ifSubdivide {
        __device__ bool operator()(const OctNode& x) {
            //return (x.children[0] == -1) && (x.hasTriangle || x.hasIntersection);
            return (x.children[0] == -1 && x.children[1] == -1 && x.children[2] == -1 && x.children[3] == -1 && x.children[4] == -1 && x.children[5] == -1 && x.children[6] == -1 && x.children[7] == -1) && (x.hasTriangle || x.hasIntersection);

        }
    };

	namespace device {
        __device__ __constant__ int maxDepth = MAX_DEPTH_OCTREE;	// Octree������

		__device__ __constant__ int decodeOffset_1 = (1 << (MAX_DEPTH_OCTREE + 1));

		__device__ __constant__ int decodeOffset_2 = (1 << (2 * (MAX_DEPTH_OCTREE + 1)));

        __device__ __constant__ int childrenVertexKind[8] = { 0, 1, 3, 2, 4, 5, 7, 6 };

        __device__ __constant__ int maxIntValue = 0x7fffffff;		// ���intֵ

        __device__ __constant__ float eps = EPSILON;

        __device__ __constant__ int edgeVertex[12][2] = { {0,1}, {2,3}, {4,5}, {6,7}, {0,3}, {1,2},
                                                          {4,7}, {5,6}, {0,4}, {1,5}, {3,7}, {2,6} };
        
        // ������8�����㣬2^8 = 256
        __constant__ int trianglesCount[256] = { 0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2,
                                                 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3,
                                                 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 
                                                 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2,
                                                 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3,
                                                 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4,
                                                 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2,
                                                 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1,
                                                 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3,
                                                 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2,
                                                 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4,
                                                 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1,
                                                 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2,
                                                 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1,
                                                 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1,
                                                 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0 };

        // Marching Cube�����β��ұ�
        __device__ __constant__ int triangles[256][16] = {
        {  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,   5,   8,   5,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,   1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,  11,   1,   9,   1,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,  11,   8,  11,   1,   8,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   0,  10,   0,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   9,  10,   9,   5,  10,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,  10,   4,  11,   4,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,  10,   8,  11,   8,   0,  11,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,  11,  10,   9,  10,   4,   9,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,  11,   8,  11,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,   2,   8,   5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   4,   6,   9,   5,   6,   2,   9,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   5,  11,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   5,  11,   6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,   2,   8,   9,  11,   1,   9,   1,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,  11,   2,   2,  11,   1,   2,   1,   6,   6,   1,   4,  -1,  -1,  -1,  -1},
        {   1,  10,   4,   2,   8,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   0,   1,   6,   2,   1,  10,   6,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   4,   1,  10,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   2,   9,   5,   6,   2,   5,   1,   6,   1,  10,   6,  -1,  -1,  -1,  -1},
        {   2,   8,   6,   4,   5,  11,   4,  11,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   2,   0,   6,   2,   5,  11,   6,   5,  10,   6,  11,  -1,  -1,  -1,  -1},
        {   9,  11,  10,   9,  10,   4,   9,   4,   0,   8,   6,   2,  -1,  -1,  -1,  -1},
        {   9,  11,   2,   2,  11,   6,  10,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   9,   2,   4,   8,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   2,   7,   0,   7,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   5,   4,   2,   7,   4,   8,   2,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   9,   2,   5,  11,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   5,  11,   0,   4,   8,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   0,   2,   1,   2,   7,   1,   7,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   7,  11,   1,   2,   7,   1,   4,   2,   4,   8,   2,  -1,  -1,  -1,  -1},
        {   4,   1,  10,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   9,   2,   0,   1,  10,   0,  10,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   1,  10,   2,   7,   5,   0,   2,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,  10,   8,   1,  10,   2,   7,   1,   2,   5,   1,   7,  -1,  -1,  -1,  -1},
        {   7,   9,   2,  10,   4,   5,  11,  10,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,  10,   8,  11,   8,   0,  11,   0,   5,   9,   2,   7,  -1,  -1,  -1,  -1},
        {  11,  10,   7,   7,  10,   4,   7,   4,   2,   2,   4,   0,  -1,  -1,  -1,  -1},
        {  11,  10,   7,   7,  10,   2,   8,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   9,   8,   6,   7,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   6,   7,   0,   4,   7,   9,   0,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,   7,   5,   8,   6,   5,   0,   8,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   6,   7,   5,   4,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,  11,   1,   8,   6,   7,   9,   8,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   6,   7,   0,   4,   7,   9,   0,   7,  11,   1,   5,  -1,  -1,  -1,  -1},
        {   8,   1,   0,  11,   1,   8,   6,  11,   8,   7,  11,   6,  -1,  -1,  -1,  -1},
        {  11,   6,   7,   1,   6,  11,   6,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,  10,   4,   6,   7,   9,   6,   9,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   1,   9,   9,   1,  10,   9,  10,   7,   7,  10,   6,  -1,  -1,  -1,  -1},
        {   6,   7,   5,   8,   6,   5,   0,   8,   5,   1,  10,   4,  -1,  -1,  -1,  -1},
        {   1,   7,   5,  10,   7,   1,   7,  10,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,  10,   4,  11,   4,   5,   7,   9,   8,   6,   7,   8,  -1,  -1,  -1,  -1},
        {   0,   6,   9,   9,   6,   7,   6,   0,   5,   5,  11,  10,   5,  10,   6,  -1},
        {   8,   7,   0,   6,   7,   8,   4,   0,   7,  11,  10,   4,   7,  11,   4,  -1},
        {  11,  10,   6,  11,   6,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   5,   0,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   7,   3,   4,   8,   9,   5,   4,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   1,   5,   3,   5,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,   7,   3,   1,   5,   7,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   1,   0,   3,   0,   9,   3,   9,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   8,   9,   4,   8,   7,   3,   4,   7,   1,   4,   3,  -1,  -1,  -1,  -1},
        {   1,  10,   4,   3,  11,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,  11,   7,   8,   0,   1,  10,   8,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   1,  10,   5,   0,   9,  11,   7,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   9,  10,   9,   5,  10,   5,   1,  11,   7,   3,  -1,  -1,  -1,  -1},
        {   4,   5,   7,   4,   7,   3,   4,   3,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   3,   3,   8,   0,   3,   0,   7,   7,   0,   5,  -1,  -1,  -1,  -1},
        {   4,   3,  10,   4,   7,   3,   4,   0,   7,   0,   9,   7,  -1,  -1,  -1,  -1},
        {  10,   8,   3,   3,   8,   7,   9,   7,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   7,   3,   8,   6,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   7,   3,   2,   0,   4,   2,   4,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   7,   3,   8,   6,   2,   5,   0,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   4,   6,   9,   5,   6,   2,   9,   6,   3,  11,   7,  -1,  -1,  -1,  -1},
        {   8,   6,   2,   3,   1,   5,   3,   5,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   1,   5,   3,   5,   7,   6,   2,   0,   4,   6,   0,  -1,  -1,  -1,  -1},
        {   3,   1,   0,   3,   0,   9,   3,   9,   7,   2,   8,   6,  -1,  -1,  -1,  -1},
        {   9,   4,   2,   2,   4,   6,   4,   9,   7,   7,   3,   1,   7,   1,   4,  -1},
        {   8,   6,   2,  11,   7,   3,   4,   1,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   0,   1,   6,   2,   1,  10,   6,   1,  11,   7,   3,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   4,   1,  10,   8,   6,   2,  11,   7,   3,  -1,  -1,  -1,  -1},
        {  11,   7,   3,   5,   2,   9,   5,   6,   2,   5,   1,   6,   1,  10,   6,  -1},
        {   4,   5,   7,   4,   7,   3,   4,   3,  10,   6,   2,   8,  -1,  -1,  -1,  -1},
        {  10,   5,   3,   3,   5,   7,   5,  10,   6,   6,   2,   0,   6,   0,   5,  -1},
        {   8,   6,   2,   4,   3,  10,   4,   7,   3,   4,   0,   7,   0,   9,   7,  -1},
        {   9,   7,  10,  10,   7,   3,  10,   6,   9,   6,   2,   9,  -1,  -1,  -1,  -1},
        {   3,  11,   9,   2,   3,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   8,   0,   2,   3,  11,   2,  11,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   2,   3,   0,   3,  11,   0,  11,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   3,   8,   8,   3,  11,   8,  11,   4,   4,  11,   5,  -1,  -1,  -1,  -1},
        {   2,   3,   1,   2,   1,   5,   2,   5,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   3,   1,   2,   1,   5,   2,   5,   9,   0,   4,   8,  -1,  -1,  -1,  -1},
        {   0,   2,   3,   0,   3,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   3,   8,   8,   3,   4,   1,   4,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,  10,   4,   9,   2,   3,  11,   9,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   0,  10,   0,   1,   3,  11,   9,   2,   3,   9,  -1,  -1,  -1,  -1},
        {   0,   2,   3,   0,   3,  11,   0,  11,   5,   1,  10,   4,  -1,  -1,  -1,  -1},
        {   5,   2,  11,  11,   2,   3,   2,   5,   1,   1,  10,   8,   1,   8,   2,  -1},
        {  10,   2,   3,   9,   2,  10,   4,   9,  10,   5,   9,   4,  -1,  -1,  -1,  -1},
        {   5,  10,   0,   0,  10,   8,  10,   5,   9,   9,   2,   3,   9,   3,  10,  -1},
        {   0,   2,   4,   4,   2,  10,   3,  10,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   8,   2,  10,   2,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   9,   8,   3,  11,   8,   6,   3,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,  11,   9,   3,  11,   0,   4,   3,   0,   6,   3,   4,  -1,  -1,  -1,  -1},
        {  11,   5,   3,   5,   0,   3,   0,   6,   3,   0,   8,   6,  -1,  -1,  -1,  -1},
        {   3,   4,   6,  11,   4,   3,   4,  11,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   1,   6,   6,   1,   5,   6,   5,   8,   8,   5,   9,  -1,  -1,  -1,  -1},
        {   0,   6,   9,   4,   6,   0,   5,   9,   6,   3,   1,   5,   6,   3,   5,  -1},
        {   3,   1,   6,   6,   1,   8,   0,   8,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   1,   4,   3,   4,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   9,   8,   3,  11,   8,   6,   3,   8,   4,   1,  10,  -1,  -1,  -1,  -1},
        {   3,   9,   6,  11,   9,   3,  10,   6,   9,   0,   1,  10,   9,   0,  10,  -1},
        {   4,   1,  10,  11,   5,   3,   5,   0,   3,   0,   6,   3,   0,   8,   6,  -1},
        {   5,  10,   6,   1,  10,   5,   6,  11,   5,   6,   3,  11,  -1,  -1,  -1,  -1},
        {  10,   5,   3,   4,   5,  10,   6,   3,   5,   9,   8,   6,   5,   9,   6,  -1},
        {   6,   3,  10,   9,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,  10,   0,   0,  10,   4,   0,   8,   3,   8,   6,   3,  -1,  -1,  -1,  -1},
        {   6,   3,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   6,  10,   0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   6,  10,   8,   9,   5,   8,   5,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   1,   5,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,   1,   5,  11,  10,   3,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   0,   9,  11,   1,   0,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,  11,   8,  11,   1,   8,   1,   4,  10,   3,   6,  -1,  -1,  -1,  -1},
        {   4,   1,   3,   6,   4,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   1,   3,   8,   0,   3,   6,   8,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   3,   6,   4,   1,   3,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,   6,   6,   9,   5,   6,   5,   3,   3,   5,   1,  -1,  -1,  -1,  -1},
        {   6,   4,   5,   6,   5,  11,   6,  11,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   6,   8,   0,   3,   6,   0,   5,   3,   5,  11,   3,  -1,  -1,  -1,  -1},
        {   3,   9,  11,   0,   9,   3,   6,   0,   3,   4,   0,   6,  -1,  -1,  -1,  -1},
        {   8,   9,   6,   6,   9,   3,  11,   3,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   2,   0,  10,   3,   0,   4,  10,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   8,  10,   3,   8,   3,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   3,   2,  10,   3,   9,   5,  10,   9,   4,  10,   5,  -1,  -1,  -1,  -1},
        {  11,   1,   5,   2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   2,   0,  10,   3,   0,   4,  10,   0,   5,  11,   1,  -1,  -1,  -1,  -1},
        {   9,  11,   1,   9,   1,   0,   2,   8,  10,   3,   2,  10,  -1,  -1,  -1,  -1},
        {  10,   2,   4,   3,   2,  10,   1,   4,   2,   9,  11,   1,   2,   9,   1,  -1},
        {   1,   3,   2,   4,   1,   2,   8,   4,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   1,   3,   2,   0,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   3,   2,   4,   1,   2,   8,   4,   2,   9,   5,   0,  -1,  -1,  -1,  -1},
        {   9,   3,   2,   5,   3,   9,   3,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   2,  11,  11,   2,   8,  11,   8,   5,   5,   8,   4,  -1,  -1,  -1,  -1},
        {   5,   2,   0,  11,   2,   5,   2,  11,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   3,   8,   8,   3,   2,   3,   4,   0,   0,   9,  11,   0,  11,   3,  -1},
        {   9,  11,   3,   9,   3,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   9,   2,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   2,   7,  10,   3,   6,   0,   4,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   7,   5,   0,   7,   0,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   5,   4,   2,   7,   4,   8,   2,   4,  10,   3,   6,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   9,   2,   7,   1,   5,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   9,   2,   7,   1,   5,  11,   0,   4,   8,  -1,  -1,  -1,  -1},
        {   1,   0,   2,   1,   2,   7,   1,   7,  11,   3,   6,  10,  -1,  -1,  -1,  -1},
        {  10,   3,   6,   1,   7,  11,   1,   2,   7,   1,   4,   2,   4,   8,   2,  -1},
        {   9,   2,   7,   6,   4,   1,   6,   1,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   1,   3,   8,   0,   3,   6,   8,   3,   7,   9,   2,  -1,  -1,  -1,  -1},
        {   0,   2,   7,   0,   7,   5,   4,   1,   3,   6,   4,   3,  -1,  -1,  -1,  -1},
        {   2,   5,   8,   7,   5,   2,   6,   8,   5,   1,   3,   6,   5,   1,   6,  -1},
        {   6,   4,   5,   6,   5,  11,   6,  11,   3,   7,   9,   2,  -1,  -1,  -1,  -1},
        {   9,   2,   7,   0,   6,   8,   0,   3,   6,   0,   5,   3,   5,  11,   3,  -1},
        {   3,   4,  11,   6,   4,   3,   7,  11,   4,   0,   2,   7,   4,   0,   7,  -1},
        {  11,   3,   8,   8,   3,   6,   8,   2,  11,   2,   7,  11,  -1,  -1,  -1,  -1},
        {   9,   8,  10,   7,   9,  10,   3,   7,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   0,   7,   0,   4,   7,   4,   3,   7,   4,  10,   3,  -1,  -1,  -1,  -1},
        {   8,  10,   0,   0,  10,   3,   0,   3,   5,   5,   3,   7,  -1,  -1,  -1,  -1},
        {  10,   5,   4,   3,   5,  10,   5,   3,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   8,  10,   7,   9,  10,   3,   7,  10,   1,   5,  11,  -1,  -1,  -1,  -1},
        {   1,   5,  11,   9,   0,   7,   0,   4,   7,   4,   3,   7,   4,  10,   3,  -1},
        {  11,   0,   7,   1,   0,  11,   3,   7,   0,   8,  10,   3,   0,   8,   3,  -1},
        {   7,   1,   4,  11,   1,   7,   4,   3,   7,   4,  10,   3,  -1,  -1,  -1,  -1},
        {   4,   9,   8,   7,   9,   4,   1,   7,   4,   3,   7,   1,  -1,  -1,  -1,  -1},
        {   7,   1,   3,   9,   1,   7,   1,   9,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   7,   0,   0,   7,   5,   7,   8,   4,   4,   1,   3,   4,   3,   7,  -1},
        {   5,   1,   3,   7,   5,   3,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   4,  11,  11,   4,   5,   4,   3,   7,   7,   9,   8,   7,   8,   4,  -1},
        {   3,   9,   0,   7,   9,   3,   0,  11,   3,   0,   5,  11,  -1,  -1,  -1,  -1},
        {   3,   7,  11,   8,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   3,   7,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,   4,   8,  10,  11,   7,  10,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   5,   0,   6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,   5,   8,   5,   4,   6,  10,  11,   7,   6,  11,  -1,  -1,  -1,  -1},
        {   5,   7,   6,   5,   6,  10,   5,  10,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   7,   6,   5,   6,  10,   5,  10,   1,   4,   8,   0,  -1,  -1,  -1,  -1},
        {   1,   0,  10,  10,   0,   9,  10,   9,   6,   6,   9,   7,  -1,  -1,  -1,  -1},
        {   1,   7,  10,  10,   7,   6,   7,   1,   4,   4,   8,   9,   4,   9,   7,  -1},
        {   7,   6,   4,   7,   4,   1,   7,   1,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   0,   1,   8,   0,  11,   7,   8,  11,   6,   8,   7,  -1,  -1,  -1,  -1},
        {   7,   6,   4,   7,   4,   1,   7,   1,  11,   5,   0,   9,  -1,  -1,  -1,  -1},
        {  11,   6,   1,   7,   6,  11,   5,   1,   6,   8,   9,   5,   6,   8,   5,  -1},
        {   4,   5,   7,   4,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   7,   0,   0,   7,   8,   6,   8,   7,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   6,   9,   9,   6,   0,   4,   0,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   9,   7,   8,   7,   6,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,  10,  11,   2,   8,  11,   7,   2,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,  11,   4,   4,  11,   7,   4,   7,   0,   0,   7,   2,  -1,  -1,  -1,  -1},
        {   8,  10,  11,   2,   8,  11,   7,   2,  11,   5,   0,   9,  -1,  -1,  -1,  -1},
        {   9,   4,   2,   5,   4,   9,   7,   2,   4,  10,  11,   7,   4,  10,   7,  -1},
        {   1,   8,  10,   2,   8,   1,   5,   2,   1,   7,   2,   5,  -1,  -1,  -1,  -1},
        {   1,   7,  10,   5,   7,   1,   4,  10,   7,   2,   0,   4,   7,   2,   4,  -1},
        {   7,   1,   9,   9,   1,   0,   1,   7,   2,   2,   8,  10,   2,  10,   1,  -1},
        {   7,   2,   9,  10,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   4,   2,   4,   1,   2,   1,   7,   2,   1,  11,   7,  -1,  -1,  -1,  -1},
        {  11,   0,   1,   7,   0,  11,   0,   7,   2,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   0,   9,   8,   4,   2,   4,   1,   2,   1,   7,   2,   1,  11,   7,  -1},
        {   2,   5,   1,   9,   5,   2,   1,   7,   2,   1,  11,   7,  -1,  -1,  -1,  -1},
        {   4,   5,   8,   8,   5,   2,   7,   2,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   2,   0,   5,   7,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   7,   2,   4,   4,   2,   8,   4,   0,   7,   0,   9,   7,  -1,  -1,  -1,  -1},
        {   7,   2,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,  11,   9,   6,  10,   9,   2,   6,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,  11,   9,   6,  10,   9,   2,   6,   9,   0,   4,   8,  -1,  -1,  -1,  -1},
        {   5,  10,  11,   6,  10,   5,   0,   6,   5,   2,   6,   0,  -1,  -1,  -1,  -1},
        {   2,   5,   8,   8,   5,   4,   5,   2,   6,   6,  10,  11,   6,  11,   5,  -1},
        {  10,   1,   6,   1,   5,   6,   5,   2,   6,   5,   9,   2,  -1,  -1,  -1,  -1},
        {   0,   4,   8,  10,   1,   6,   1,   5,   6,   5,   2,   6,   5,   9,   2,  -1},
        {   1,   0,  10,  10,   0,   6,   2,   6,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   6,   1,   1,   6,  10,   1,   4,   2,   4,   8,   2,  -1,  -1,  -1,  -1},
        {  11,   9,   1,   1,   9,   2,   1,   2,   4,   4,   2,   6,  -1,  -1,  -1,  -1},
        {   8,   1,   6,   0,   1,   8,   2,   6,   1,  11,   9,   2,   1,  11,   2,  -1},
        {  11,   6,   1,   1,   6,   4,   6,  11,   5,   5,   0,   2,   5,   2,   6,  -1},
        {   2,   6,   8,  11,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   6,   4,   2,   2,   4,   9,   5,   9,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   9,   6,   6,   9,   2,   6,   8,   5,   8,   0,   5,  -1,  -1,  -1,  -1},
        {   0,   2,   6,   0,   6,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   2,   6,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,  10,  11,   9,   8,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   0,  11,   9,   4,  11,   0,  11,   4,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,  10,  11,   0,  10,   5,  10,   0,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,  10,  11,   5,   4,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,   8,  10,   5,   8,   1,   8,   5,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   4,  10,   0,   4,   9,  10,   5,   9,  10,   1,   5,  -1,  -1,  -1,  -1},
        {   0,   8,  10,   1,   0,  10,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  10,   1,   4,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   4,   9,   8,   1,   9,   4,   9,   1,  11,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   1,  11,   9,   0,   1,   9,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  11,   0,   8,   5,   0,  11,   8,   1,  11,   8,   4,   1,  -1,  -1,  -1,  -1},
        {  11,   5,   1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   5,   9,   8,   4,   5,   8,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   9,   0,   5,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {   8,   4,   0,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1},
        {  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1,  -1} };
        
        __device__ __constant__ int faceEdges[6][4] = { {4,  6,  8,  10},
                                                        {5,  7,  9,  11},
                                                        {0,  2,  8,  9 },
                                                        {1,  3,  10, 11},
                                                        {0,  1,  4,  5 },
                                                        {2,  3,  6,  7 } };

        __device__ __constant__ int LUTparent[8][27] = {
        {0,1,1,3,4,4,3,4,4,9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13},
        {1,1,2,4,4,5,4,4,5,10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14},
        {3,4,4,3,4,4,6,7,7,12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16},
        {4,4,5,4,4,5,7,7,8,13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17},
        {9,10,10,12,13,13,12,13,13,9,10,10,12,13,13,12,13,13,18,19,19,21,22,22,21,22,22},
        {10,10,11,13,13,14,13,13,14,10,10,11,13,13,14,13,13,14,19,19,20,22,22,23,22,22,23},
        {12,13,13,12,13,13,15,16,16,12,13,13,12,13,13,15,16,16,21,22,22,21,22,22,24,25,25},
        {13,13,14,13,13,14,16,16,17,13,13,14,13,13,14,16,16,17,22,22,23,22,22,23,25,25,26}
        };
        __device__ __constant__ int LUTchild[8][27] = {
                {7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7},
                {6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6},
                {5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5},
                {4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4},
                {3,2,3,1,0,1,3,2,3,7,6,7,5,4,5,7,6,7,3,2,3,1,0,1,3,2,3},
                {2,3,2,0,1,0,2,3,2,6,7,6,4,5,4,6,7,6,2,3,2,0,1,0,2,3,2},
                {1,0,1,3,2,3,1,0,1,5,4,5,7,6,7,5,4,5,1,0,1,3,2,3,1,0,1},
                {0,1,0,2,3,2,0,1,0,4,5,4,6,7,6,4,5,4,0,1,0,2,3,2,0,1,0}
        };
	}
}
__global__ void SparseSurfelFusion::device::ComputeVertexImplicitFunctionValueKernel(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunctions, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const unsigned int VertexArraySize, const float isoValue, float* vvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= VertexArraySize)	return;
    VertexNode nowVertex = VertexArray[idx];
    int depth = nowVertex.depth;
    float val = 0.0f;
    int exceedChildrenId = childrenVertexKind[nowVertex.vertexKind];
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        while (nowNode != -1) {
            for (int i = 0; i < 27; i++) {
                int neighbor = NodeArray[nowNode].neighs[i];
                if (neighbor != -1) {
                    int idxO[3];
                    int encode_idx = encodeNodeIndexInFunction[neighbor];
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = BaseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = BaseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = BaseFunctions[idxO[2]];

                    val += dx[neighbor] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            nowNode = NodeArray[nowNode].parent;
        }
        nowNode = nowVertex.ownerNodeIdx;
        while (depth < device::maxDepth) {
            depth++;
            nowNode = NodeArray[nowNode].children[exceedChildrenId];
            if (nowNode == -1) break;
            for (int i = 0; i < 27; i++) {
                int neighbor = NodeArray[nowNode].neighs[i];
                if (neighbor != -1) {
                    int idxO[3];
                    int encode_idx = encodeNodeIndexInFunction[neighbor];
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = BaseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = BaseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = BaseFunctions[idxO[2]];

                    val += dx[neighbor] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
        }
    }
    vvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::generateVertexNumsKernel(DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int EdgeArraySize, int* vexNums, bool* markValidVertex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= EdgeArraySize)	return;
    EdgeNode nowEdge = EdgeArray[idx];
    int owner = nowEdge.ownerNodeIdx;
    int kind = nowEdge.edgeKind;
    int index[2];
    index[0] = device::edgeVertex[kind][0];
    index[1] = device::edgeVertex[kind][1];

    int v1 = NodeArray[owner].vertices[index[0]] - 1;
    int v2 = NodeArray[owner].vertices[index[1]] - 1;
    if (vvalue[v1] * vvalue[v2] <= 0) {
        vexNums[idx] = 1;
        markValidVertex[idx] = true;
    }
    else {
        vexNums[idx] = 0;
        markValidVertex[idx] = false;
    }
}

__global__ void SparseSurfelFusion::device::generateTriangleNumsKernel(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, int* triNums, int* cubeCatagory)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    OctNode currentNode = NodeArray[offset];    // ��ǰ����ڵ�
    int currentCubeCatagory = 0;                // ����������
    for (int i = 0; i < 8; i++) {
        if (vvalue[currentNode.vertices[i] - 1] < 0) {
            currentCubeCatagory |= 1 << i;
        }
    }
    triNums[idx] = device::trianglesCount[currentCubeCatagory];
    cubeCatagory[idx] = currentCubeCatagory;
}

__global__ void SparseSurfelFusion::device::generateIntersectionPoint(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<float> vvalue, const EdgeNode* validEdgeArray, const int* validVexAddress, const unsigned int validEdgeArraySize, Point3D<float>* VertexBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= validEdgeArraySize)	return;
    int owner = validEdgeArray[idx].ownerNodeIdx;
    int kind = validEdgeArray[idx].edgeKind;
    int orientation = kind >> 2;

    int index[2];

    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];

    int v1 = NodeArray[owner].vertices[index[0]] - 1;
    int v2 = NodeArray[owner].vertices[index[1]] - 1;
    Point3D<float> p1 = VertexArray[v1].pos; 
    Point3D<float> p2 = VertexArray[v2].pos;
    float f1 = vvalue[v1];
    float f2 = vvalue[v2];
    Point3D<float> isoPoint;
    interpolatePoint(p1, p2, orientation, f1, f2, isoPoint);
    VertexBuffer[validVexAddress[idx]] = isoPoint;
}

__device__ void SparseSurfelFusion::device::interpolatePoint(const Point3D<float>& p1, const Point3D<float>& p2, const int& dim, const float& v1, const float& v2, Point3D<float>& out)
{
    for (int i = 0; i < 3; i++) {
        if (i != dim) {
            out.coords[i] = p1.coords[i];
        }
    }
    float pivot = v1 / (v1 - v2);
    float anotherPivot = 1 - pivot;
    out.coords[dim] = p2.coords[dim] * pivot + p1.coords[dim] * anotherPivot;
}

__global__ void SparseSurfelFusion::device::generateTrianglePos(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<FaceNode> FaceArray, DeviceArrayView<int> triNums, DeviceArrayView<int> cubeCatagory, DeviceArrayView<int> vexAddress, DeviceArrayView<int> triAddress, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, TriangleIndex* TriangleBuffer, int* hasSurfaceIntersection)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    OctNode currentNode = NodeArray[offset];
    int currentTriNum = triNums[idx];
    int currentCubeCatagory = cubeCatagory[idx];
    int currentTriangleBufferStart = triAddress[idx];
    int edgeHasVertex[12] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

    //if (idx % 1000 == 0) printf("index = %d   TriNum = %d\n", idx, currentTriNum);

    for (int i = 0; i < currentTriNum; i++) {
        int edgeIdx[3];
        edgeIdx[0] = device::triangles[currentCubeCatagory][3 * i];
        edgeIdx[1] = device::triangles[currentCubeCatagory][3 * i + 1];
        edgeIdx[2] = device::triangles[currentCubeCatagory][3 * i + 2];

        edgeHasVertex[edgeIdx[0]] = 1;
        edgeHasVertex[edgeIdx[1]] = 1;
        edgeHasVertex[edgeIdx[2]] = 1;

        int vertexIdx[3];
        vertexIdx[0] = vexAddress[currentNode.edges[edgeIdx[0]] - 1];
        vertexIdx[1] = vexAddress[currentNode.edges[edgeIdx[1]] - 1];
        vertexIdx[2] = vexAddress[currentNode.edges[edgeIdx[2]] - 1];

        TriangleBuffer[currentTriangleBufferStart + i].idx[0] = vertexIdx[0];
        TriangleBuffer[currentTriangleBufferStart + i].idx[1] = vertexIdx[1];
        TriangleBuffer[currentTriangleBufferStart + i].idx[2] = vertexIdx[2];
    }
    int currentFace;
    int parentNodeIndex;
    for (int i = 0; i < 6; i++) {
        int mark = 0;              // ��¼�Ƿ����Surface-Edge Intersections(����ཻ)
        for (int j = 0; j < 4; j++) {
            mark |= edgeHasVertex[device::faceEdges[i][j]];
        }
        if (mark == 1) {
            parentNodeIndex = NodeArray[offset].parent;
            currentFace = currentNode.faces[i] - 1;
            hasSurfaceIntersection[currentFace] = 1;
            while (FaceArray[currentFace].hasParentFace != -1) {
                currentFace = NodeArray[parentNodeIndex].faces[i] - 1;
                parentNodeIndex = NodeArray[parentNodeIndex].parent;
                hasSurfaceIntersection[currentFace] = 1;
            }
        }
    }
}

__global__ void SparseSurfelFusion::device::generateSubdivideTrianglePos(const EasyOctNode* SubdivideArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, const int* SubdivideTriNums, const int* SubdivideCubeCatagory, const int* SubdivideVexAddress, const int* SubdivideTriAddress, TriangleIndex* SubdivideTriangleBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int nowTriNum = SubdivideTriNums[idx];
    int nowCubeCatagory = SubdivideCubeCatagory[idx];
    int nowTriangleBufferStart = SubdivideTriAddress[idx];
    //if (flag == true && nowTriNum != 0) printf("idx = %d   nowTriNum = %d   nowTriangleBufferStart = %d\n", idx, nowTriNum, nowTriangleBufferStart);
    for (int i = 0; i < nowTriNum; i++) {
        int edgeIdx[3];
        edgeIdx[0] = triangles[nowCubeCatagory][3 * i];
        edgeIdx[1] = triangles[nowCubeCatagory][3 * i + 1];
        edgeIdx[2] = triangles[nowCubeCatagory][3 * i + 2];

        int vertexIdx[3];
        vertexIdx[0] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[0]] - 1];
        vertexIdx[1] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[1]] - 1];
        vertexIdx[2] = SubdivideVexAddress[SubdivideArray[offset].edges[edgeIdx[2]] - 1];
        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[0] = vertexIdx[0];
        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[1] = vertexIdx[1];
        SubdivideTriangleBuffer[nowTriangleBufferStart + i].idx[2] = vertexIdx[2];
        //if (flag == true)    printf("idx = %d   nowTriangleBufferStart = %d   vertexIdx[%d] = (%d, %d, %d)\n", idx, nowTriangleBufferStart, nowTriangleBufferStart + i, vertexIdx[0], vertexIdx[1], vertexIdx[2]);

    }
}

__global__ void SparseSurfelFusion::device::ProcessLeafNodesAtOtherDepth(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<float> vvalue, const unsigned int OtherDepthNodeCount, const int* hasSurfaceIntersection, OctNode* NodeArray, bool* markValidSubdividedNode)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= OtherDepthNodeCount)	return;
    OctNode currentNode = NodeArray[idx];
    int hasTri = 0;
    int sign = (vvalue[currentNode.vertices[0] - 1] < 0) ? -1 : 1;
    for (int i = 1; i < 8; i++) {
        if (sign * vvalue[currentNode.vertices[i] - 1] < 0) {
            hasTri = 1;
            break;
        }
    }

    NodeArray[idx].hasTriangle = hasTri;

    int hasIntersection = 0;
    for (int i = 0; i < 6; i++) {
        if (hasSurfaceIntersection[currentNode.faces[i] - 1]) {
            hasIntersection = 1;
            break;
        }
    }
    NodeArray[idx].hasIntersection = hasIntersection;


    if ((NodeArray[idx].children[0] == -1) && (hasTri || hasIntersection)) {
        markValidSubdividedNode[idx] = true;
    }
    else {
        markValidSubdividedNode[idx] = false;
    }
}

__global__ void SparseSurfelFusion::device::precomputeSubdivideDepth(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<unsigned int> DepthBuffer, const int SubdivideNum, int* SubdivideDepthBuffer, int* SubdivideDepthNum)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideNum)	return;
    int nodeIndex = SubdivideNode[idx].neighs[13];
    int depth = DepthBuffer[nodeIndex];
    SubdivideDepthBuffer[idx] = depth;
    SubdivideDepthNum[idx + depth * SubdivideNum] = 1;
}

__global__ void SparseSurfelFusion::device::singleRebuildArray(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<int> SubdivideDepthBuffer, const unsigned int iterRound, const unsigned int NodeArraySize, const unsigned int SubdivideArraySize, EasyOctNode* SubdivideArray, int* SubdivideArrayDepthBuffer, Point3D<float>* SubdivideArrayCenterBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideArraySize)	return;
    int rootId = SubdivideNode[iterRound].neighs[13];
    int rootDepth = SubdivideDepthBuffer[iterRound];
    int rootKey = SubdivideNode[iterRound].key;
    int thisNodeDepth = getSubdivideDepth(rootDepth, idx);
    int relativeDepth = thisNodeDepth - rootDepth;
    int idxOffset = idx - (powf(8, relativeDepth) - 1) / 7;

    if (thisNodeDepth < maxDepth) {
        int nextDepthAddress = (powf(8, relativeDepth + 1) - 1) / 7;
        for (int k = 0; k < 8; ++k) {
            SubdivideArray[idx].children[k] = NodeArraySize + nextDepthAddress + (idxOffset << 3) + k;
        }
    }
    else {
        for (int k = 0; k < 8; ++k) {
            SubdivideArray[idx].children[k] = -1;
        }
    }

    if (idx != 0) {
        int parentDepthAddress = (powf(8, relativeDepth - 1) - 1) / 7;
        SubdivideArray[idx].parent = NodeArraySize + parentDepthAddress + (idxOffset >> 3);
    }

    int thisKey = rootKey;
    thisKey |= (idxOffset) << (3 * (maxDepth - thisNodeDepth));
    SubdivideArray[idx].key = thisKey;

    SubdivideArrayDepthBuffer[idx] = thisNodeDepth;
    Point3D<float> thisNodeCenter;
    getNodeCenterAllDepth(thisKey, thisNodeDepth, thisNodeCenter);
    SubdivideArrayCenterBuffer[idx] = thisNodeCenter;
}

__device__ int SparseSurfelFusion::device::getSubdivideDepth(const int& rootDepth, const int& idx)
{
    int up = idx * 7 + 1;
    int base = 8;
    int relativeDepth = 0;
    while (base <= up) {
        relativeDepth++;
        base <<= 3;
    }
    return rootDepth + relativeDepth;
}

__device__ void SparseSurfelFusion::device::getNodeCenterAllDepth(const int& key, const int& currentDepth, Point3D<float>& center)
{
    center.coords[0] = float(0.5);
    center.coords[1] = float(0.5);
    center.coords[2] = float(0.5);
    float Width = 0.25f;
    for (int i = device::maxDepth - 1; i >= (device::maxDepth - currentDepth); --i) {
        if ((key >> (3 * i + 2)) & 1) center.coords[0] += Width;
        else center.coords[0] -= Width;

        if ((key >> (3 * i + 1)) & 1) center.coords[1] += Width;
        else center.coords[1] -= Width;

        if ((key >> (3 * i)) & 1) center.coords[2] += Width;
        else center.coords[2] -= Width;

        Width /= 2;
    }
}

__global__ void SparseSurfelFusion::device::computeRebuildNeighbor(DeviceArrayView<OctNode> NodeArray, const unsigned int currentLevelOffset, const unsigned int currentLevelNodesCount, const unsigned int NodeArraySize, const unsigned int depth, EasyOctNode* SubdivideArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= currentLevelNodesCount)	return;
    const unsigned int offset = currentLevelOffset + idx;
    for (int i = 0; i < 27; i++) {
        int sonKey = (SubdivideArray[offset].key >> (3 * (device::maxDepth - depth))) & 7;
        int parentIdx = SubdivideArray[offset].parent;
        int neighParent;
        if (parentIdx < NodeArraySize) {
            neighParent = NodeArray[parentIdx].neighs[device::LUTparent[sonKey][i]];
        }
        else {
            neighParent = SubdivideArray[parentIdx - NodeArraySize].neighs[device::LUTparent[sonKey][i]];
        }
        if (neighParent != -1) {
            if (neighParent < NodeArraySize) {
                SubdivideArray[offset].neighs[i] = NodeArray[neighParent].children[device::LUTchild[sonKey][i]];
            }
            else {
                SubdivideArray[offset].neighs[i] = SubdivideArray[neighParent - NodeArraySize].children[device::LUTchild[sonKey][i]];
            }
        }
        else {
            SubdivideArray[offset].neighs[i] = -1;
        }
    }
}

__global__ void SparseSurfelFusion::device::initSubdivideVertexOwner(const EasyOctNode* SubdivideArray, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int currentLevelOffset, const unsigned int currentLevelNodesCount, const unsigned int NodeArraySize, VertexNode* SubdividePreVertexArray, bool* markValidSubdivideVertex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= currentLevelNodesCount)	return;
    const unsigned int offset = currentLevelOffset + idx;
    int NodeOwnerKey[8] = { device::maxIntValue,device::maxIntValue, device::maxIntValue, device::maxIntValue,
                            device::maxIntValue, device::maxIntValue, device::maxIntValue, device::maxIntValue };
    int NodeOwnerIdx[8] = { -1, -1, -1, -1, -1, -1, -1, -1 };
    int depth = maxDepth;
    float halfWidth = 1.0f / (1 << (depth + 1));
    float Width = 1.0f / (1 << depth);
    float Widthsq = Width * Width;
    Point3D<float> neighCenter[27];
    int neigh[27];
#pragma unroll
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[offset].neighs[i];
        if (neigh[i] != -1 && neigh[i] >= NodeArraySize) {
            neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
        }
    }
    const Point3D<float>& nodeCenter = neighCenter[13];

    Point3D<float> vertexPos[8];
#pragma unroll
    for (int i = 0; i < 8; i++) {
        vertexPos[i].coords[0] = nodeCenter.coords[0] + (2 * (i & 1) - 1) * halfWidth;
        vertexPos[i].coords[1] = nodeCenter.coords[1] + (2 * ((i & 2) >> 1) - 1) * halfWidth;
        vertexPos[i].coords[2] = nodeCenter.coords[2] + (2 * ((i & 4) >> 2) - 1) * halfWidth;
    }

#pragma unroll

    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 27; j++) {
            if (neigh[j] != -1 && SquareDistance(vertexPos[i], neighCenter[j]) < Widthsq) {
                int neighKey;
                if (neigh[j] < NodeArraySize) continue;
                else
                    neighKey = SubdivideArray[neigh[j] - NodeArraySize].key;
                if (NodeOwnerKey[i] > neighKey) {
                    NodeOwnerKey[i] = neighKey;
                    NodeOwnerIdx[i] = neigh[j];
                }
            }
        }
    }
#pragma unroll
    for (int i = 0; i < 8; i++) {
        int vertexIdx = 8 * idx + i;
        if (NodeOwnerIdx[i] == NodeArraySize + offset) {
            SubdividePreVertexArray[vertexIdx].ownerNodeIdx = NodeOwnerIdx[i];
            SubdividePreVertexArray[vertexIdx].pos.coords[0] = vertexPos[i].coords[0];
            SubdividePreVertexArray[vertexIdx].pos.coords[1] = vertexPos[i].coords[1];
            SubdividePreVertexArray[vertexIdx].pos.coords[2] = vertexPos[i].coords[2];
            SubdividePreVertexArray[vertexIdx].vertexKind = i;
            SubdividePreVertexArray[vertexIdx].depth = depth;
            markValidSubdivideVertex[vertexIdx] = true;
            //printf("idx = %d   vertexIdx = %d\n", idx, vertexIdx);
        }
        else {
            markValidSubdivideVertex[vertexIdx] = false;
        }
    }

}

__global__ void SparseSurfelFusion::device::maintainSubdivideVertexNodePointer(DeviceArrayView<Point3D<float>> CenterBuffer, const unsigned int VertexArraySize, const unsigned int NodeArraySize, const Point3D<float>* SubdivideArrayCenterBuffer, VertexNode* VertexArray, EasyOctNode* SubdivideArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= VertexArraySize)	return;
    int owner = VertexArray[idx].ownerNodeIdx;
    float Width = 1.0f / (1 << device::maxDepth);
    float WidthSquare = Width * Width;
    Point3D<float> neighCenter[27];
    Point3D<float> vertexPos = VertexArray[idx].pos;

    int neigh[27];
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[owner - NodeArraySize].neighs[i];
    }
    for (int i = 0; i < 27; i++) {
        if (neigh[i] != -1) {
            if (neigh[i] < NodeArraySize) {
                neighCenter[i] = CenterBuffer[neigh[i]];
            }
            else {
                neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
            }
        }
    }
    int count = 0;
    for (int i = 0; i < 27; i++) {
        if (neigh[i] != -1 && SquareDistance(vertexPos, neighCenter[i]) < WidthSquare) {
            VertexArray[idx].nodes[count] = neigh[i];
            count++;
            int index = 0;
            if (neighCenter[i].coords[0] - vertexPos.coords[0] < 0) index |= 1;
            if (neighCenter[i].coords[2] - vertexPos.coords[2] < 0) index |= 4;
            if (neighCenter[i].coords[1] - vertexPos.coords[1] < 0) {
                if (index & 1) {
                    index += 1;
                }
                else {
                    index += 3;
                }
            }
            if (neigh[i] >= NodeArraySize) {
                SubdivideArray[neigh[i] - NodeArraySize].vertices[index] = idx + 1;
            }
        }
    }
}

__global__ void SparseSurfelFusion::device::initSubdivideEdgeArray(const EasyOctNode* SubdivideArray, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int NodeArraySize, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, EdgeNode* SubdividePreEdgeArray, bool* markValidSubdivideEdge)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int NodeOwnerKey[12] = { device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue,
                             device::maxIntValue, device::maxIntValue, device::maxIntValue };
    int NodeOwnerIdx[12] = { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };
    float halfWidth = 1.0f / (1 << (device::maxDepth + 1));
    float Width = 1.0f / (1 << device::maxDepth);
    float WidthSquare = Width * Width;
    Point3D<float> neighCenter[27];
    int neigh[27];
#pragma unroll
    for (int i = 0; i < 27; i++) {
        neigh[i] = SubdivideArray[offset].neighs[i];
        if (neigh[i] != -1 && neigh[i] >= NodeArraySize) {
            neighCenter[i] = SubdivideArrayCenterBuffer[neigh[i] - NodeArraySize];
        }
    }
    const Point3D<float>& nodeCenter = neighCenter[13];
    Point3D<float> edgeCenterPos[12];
    int orientation[12];
    int off[24];
#pragma unroll
    for (int i = 0; i < 12; i++) {
        orientation[i] = i >> 2;
        off[2 * i] = i & 1;
        off[2 * i + 1] = (i & 2) >> 1;
        int multi[3];
        int dim = 2 * i;
        for (int j = 0; j < 3; j++) {
            if (orientation[i] == j) {
                multi[j] = 0;
            }
            else {
                multi[j] = (2 * off[dim] - 1);
                dim++;
            }
        }
        edgeCenterPos[i].coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
        edgeCenterPos[i].coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
        edgeCenterPos[i].coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;
    }

#pragma unroll
    for (int i = 0; i < 12; i++) {
        for (int j = 0; j < 27; j++) {
            if (neigh[j] != -1 && SquareDistance(edgeCenterPos[i], neighCenter[j]) < WidthSquare) {
                int neighKey;
                if (neigh[j] < NodeArraySize) continue;
                else
                    neighKey = SubdivideArray[neigh[j] - NodeArraySize].key;
                if (NodeOwnerKey[i] > neighKey) {
                    NodeOwnerKey[i] = neighKey;
                    NodeOwnerIdx[i] = neigh[j];
                }
            }
        }
    }
#pragma unroll
    for (int i = 0; i < 12; i++) {
        int edgeIdx = 12 * idx + i;
        if (NodeOwnerIdx[i] == offset + NodeArraySize) {
            SubdividePreEdgeArray[edgeIdx].ownerNodeIdx = NodeOwnerIdx[i];
            SubdividePreEdgeArray[edgeIdx].edgeKind = i;
            markValidSubdivideEdge[edgeIdx] = true;
        }
        else {
            markValidSubdivideEdge[edgeIdx] = false;
        }
    }
}

__global__ void SparseSurfelFusion::device::maintainSubdivideEdgeNodePointer(DeviceArrayView<Point3D<float>> CenterBuffer, const Point3D<float>* SubdivideArrayCenterBuffer, const unsigned int EdgeArraySize, const unsigned int NodeArraySize, EasyOctNode* SubdivideArray, EdgeNode* EdgeArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= EdgeArraySize)	return;
    int owner = EdgeArray[idx].ownerNodeIdx;

    int depth = device::maxDepth;
    float halfWidth = 1.0f / (1 << (depth + 1));
    float Width = 1.0f / (1 << depth);
    float WidthSquare = Width * Width;

    Point3D<float> neighCenter[27];
    int neighbor[27];
    for (int i = 0; i < 27; i++) {
        neighbor[i] = SubdivideArray[owner - NodeArraySize].neighs[i];
        if (neighbor[i] != -1) {
            if (neighbor[i] < NodeArraySize)
                neighCenter[i] = CenterBuffer[neighbor[i]];
            else
                neighCenter[i] = SubdivideArrayCenterBuffer[neighbor[i] - NodeArraySize];
        }
    }

    const Point3D<float>& nodeCenter = neighCenter[13];
    Point3D<float> edgeCenterPos;
    int multi[3];
    int dim = 0;
    int orientation = EdgeArray[idx].edgeKind >> 2;
    int off[2];
    off[0] = EdgeArray[idx].edgeKind & 1;
    off[1] = (EdgeArray[idx].edgeKind & 2) >> 1;
    for (int i = 0; i < 3; i++) {
        if (orientation == i) {
            multi[i] = 0;
        }
        else {
            multi[i] = (2 * off[dim] - 1);
            dim++;
        }
    }
    edgeCenterPos.coords[0] = nodeCenter.coords[0] + multi[0] * halfWidth;
    edgeCenterPos.coords[1] = nodeCenter.coords[1] + multi[1] * halfWidth;
    edgeCenterPos.coords[2] = nodeCenter.coords[2] + multi[2] * halfWidth;

    int count = 0;
    for (int i = 0; i < 27; i++) {
        if (neighbor[i] != -1 && SquareDistance(edgeCenterPos, neighCenter[i]) < WidthSquare) {
            EdgeArray[idx].nodes[count] = neighbor[i];
            count++;
            int index = orientation << 2;
            int dim = 0;
            for (int j = 0; j < 3; j++) {
                if (orientation != j) {
                    if (neighCenter[i].coords[j] - edgeCenterPos.coords[j] < 0) index |= (1 << dim);
                    dim++;
                }
            }
            if (neighbor[i] >= NodeArraySize)
                SubdivideArray[neighbor[i] - NodeArraySize].edges[index] = idx + 1;
        }
    }
}

__forceinline__ __device__ double SparseSurfelFusion::device::SquareDistance(const Point3D<float>& p1, const Point3D<float>& p2)
{
    return (p1.coords[0] - p2.coords[0]) * (p1.coords[0] - p2.coords[0]) + (p1.coords[1] - p2.coords[1]) * (p1.coords[1] - p2.coords[1]) + (p1.coords[2] - p2.coords[2]) * (p1.coords[2] - p2.coords[2]);
}

__global__ void SparseSurfelFusion::device::computeSubdivideVertexImplicitFunctionValue(const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> dx, DeviceArrayView<int> EncodedNodeIdxInFunction, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> baseFunctions, const unsigned int NodeArraySize, const unsigned int rootId, const unsigned int SubdivideVertexArraySize, const float isoValue, float* SubdivideVvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideVertexArraySize)	return;
    VertexNode nowVertex = SubdivideVertexArray[idx];
    int depth = device::maxDepth;
    float val = 0.0f;
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        while (nowNode != -1) {
            for (int k = 0; k < 27; ++k) {
                int neigh;
                if (nowNode < NodeArraySize)
                    neigh = NodeArray[nowNode].neighs[k];
                else
                    neigh = SubdivideArray[nowNode - NodeArraySize].neighs[k];
                if (neigh != -1) {
                    if (neigh == NodeArraySize)
                        neigh = rootId;
                    int idxO[3];
                    int encode_idx;
                    if (neigh < NodeArraySize)
                        encode_idx = EncodedNodeIdxInFunction[neigh];
                    else continue;  // d_x = 0 in Subdivide space
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = baseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = baseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = baseFunctions[idxO[2]];

                    val += dx[neigh] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            if (nowNode < NodeArraySize)
                nowNode = NodeArray[nowNode].parent;
            else
                nowNode = SubdivideArray[nowNode - NodeArraySize].parent;
        }
    }
    SubdivideVvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::computeSubdivideVertexImplicitFunctionValue(const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> dx, DeviceArrayView<int> EncodedNodeIdxInFunction, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> baseFunctions, const unsigned int NodeArraySize, const int* ReplacedNodeId, const int* IsRoot, const unsigned int SubdivideVertexArraySize, const float isoValue, float* SubdivideVvalue)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideVertexArraySize)	return;
    VertexNode nowVertex = SubdivideVertexArray[idx];
    float val = 0.0f;
    int nowNode = nowVertex.ownerNodeIdx;
    if (nowNode > 0) {
        while (nowNode != -1) {
            for (int k = 0; k < 27; ++k) {
                int neigh;
                if (nowNode < NodeArraySize)
                    neigh = NodeArray[nowNode].neighs[k];
                else
                    neigh = SubdivideArray[nowNode - NodeArraySize].neighs[k];
                if (neigh != -1) {
                    if (neigh >= NodeArraySize && IsRoot[neigh - NodeArraySize])
                        neigh = ReplacedNodeId[neigh - NodeArraySize];
                    int idxO[3];
                    int encode_idx;
                    if (neigh < NodeArraySize)
                        encode_idx = EncodedNodeIdxInFunction[neigh];
                    else continue;  // d_x = 0 in Subdivide space
                    idxO[0] = encode_idx % decodeOffset_1;
                    idxO[1] = (encode_idx / decodeOffset_1) % decodeOffset_1;
                    idxO[2] = encode_idx / decodeOffset_2;

                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcX = baseFunctions[idxO[0]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcY = baseFunctions[idxO[1]];
                    ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2> funcZ = baseFunctions[idxO[2]];

                    val += dx[neigh] * value(funcX, nowVertex.pos.coords[0]) * value(funcY, nowVertex.pos.coords[1]) * value(funcZ, nowVertex.pos.coords[2]);
                }
            }
            if (nowNode < NodeArraySize) nowNode = NodeArray[nowNode].parent;
            else nowNode = SubdivideArray[nowNode - NodeArraySize].parent;
        }
    }
    SubdivideVvalue[idx] = val - isoValue;
}

__global__ void SparseSurfelFusion::device::generateSubdivideVexNums(const EdgeNode* SubdivideEdgeArray, const EasyOctNode* SubdivideArray, const unsigned int SubdivideEdgeArraySize, const unsigned int NodeArraySize, const float* SubdivideVvalue, int* SubdivideVexNums, bool* markValidSubdivedeVexNum)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideEdgeArraySize)	return;
    int owner = SubdivideEdgeArray[idx].ownerNodeIdx - NodeArraySize;   // ��ǰ�ߵ�Owner
    int kind = SubdivideEdgeArray[idx].edgeKind;                        // ��ǰ�ߵ�����
    int index[2];
    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];
    int v1 = SubdivideArray[owner].vertices[index[0]] - 1;
    int v2 = SubdivideArray[owner].vertices[index[1]] - 1;
    if (SubdivideVvalue[v1] * SubdivideVvalue[v2] <= 0) {
        SubdivideVexNums[idx] = 1;
        markValidSubdivedeVexNum[idx] = true;
    }
    else {
        markValidSubdivedeVexNum[idx] = false;
    }

}

__global__ void SparseSurfelFusion::device::generateTriNums(const EasyOctNode* SubdivideNodeArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, const float* vvalue, int* triNums, int* cubeCatagory)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DLevelNodeCount)	return;
    const unsigned int offset = DLevelOffset + idx;
    int currentCubeCatagory = 0;
    for (int i = 0; i < 8; i++) {
        if (vvalue[SubdivideNodeArray[offset].vertices[i] - 1] < 0) {
            currentCubeCatagory |= 1 << i;
        }
    }
    triNums[idx] = trianglesCount[currentCubeCatagory];
    cubeCatagory[idx] = currentCubeCatagory;
}

__global__ void SparseSurfelFusion::device::generateSubdivideIntersectionPoint(const EdgeNode* SubdivideValidEdgeArray, const VertexNode* SubdivideVertexArray, const EasyOctNode* SubdivideArray, const int* SubdivideValidVexAddress, const float* SubdivideVvalue, const unsigned int SubdivideValidEdgeArraySize, const unsigned int NodeArraySize, Point3D<float>* SubdivideVertexBuffer)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= SubdivideValidEdgeArraySize)	return;
    int owner = SubdivideValidEdgeArray[idx].ownerNodeIdx - NodeArraySize;
    int kind = SubdivideValidEdgeArray[idx].edgeKind;
    int orientation = kind >> 2;
    int index[2];

    index[0] = edgeVertex[kind][0];
    index[1] = edgeVertex[kind][1];

    int v1 = SubdivideArray[owner].vertices[index[0]] - 1;
    int v2 = SubdivideArray[owner].vertices[index[1]] - 1;
    Point3D<float> p1 = SubdivideVertexArray[v1].pos, p2 = SubdivideVertexArray[v2].pos;
    float f1 = SubdivideVvalue[v1];
    float f2 = SubdivideVvalue[v2];
    Point3D<float> isoPoint;
    device::interpolatePoint(p1, p2, orientation, f1, f2, isoPoint);
    SubdivideVertexBuffer[SubdivideValidVexAddress[idx]] = isoPoint;
}

__global__ void SparseSurfelFusion::device::initFixedDepthNums(DeviceArrayView<OctNode> SubdivideNode, DeviceArrayView<int> SubdivideDepthBuffer, const unsigned int DepthOffset, const unsigned int DepthNodeCount, int* fixedDepthNums)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= DepthNodeCount)	return;
    const unsigned int offset = DepthOffset + idx;
    int nodeNum = 1;
    for (int depth = SubdivideDepthBuffer[offset]; depth <= device::maxDepth; ++depth) {
        fixedDepthNums[(depth - 1) * DepthNodeCount + idx] = nodeNum;
        nodeNum <<= 3;  // ��8
    }
}

__global__ void SparseSurfelFusion::device::wholeRebuildArray(DeviceArrayView<OctNode> SubdivideNode, const unsigned int finerDepthStart, const unsigned int finerSubdivideNum, const unsigned int NodeArraySize, const int* SubdivideDepthBuffer, const int* depthNodeAddress_Device, const int* fixedDepthAddress, EasyOctNode* RebuildArray, int* RebuildDepthBuffer, Point3D<float>* RebuildCenterBuffer, int* ReplaceNodeId, int* IsRoot, OctNode* NodeArray)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= finerSubdivideNum)	return;
    int depthNodeAddress[MAX_DEPTH_OCTREE + 1];
    for (int i = 0; i <= maxDepth; i++) {                   // ������ݸ��죺L1����
        depthNodeAddress[i] = depthNodeAddress_Device[i];   // depthNodeAddress_Device��Global�ڴ棬������Ҫѭ����ʱ�俪����
    }
    const unsigned int offset = finerDepthStart + idx;
    int nowDepth = SubdivideDepthBuffer[offset];
    int fixedDepthOffset = fixedDepthAddress[(nowDepth - 1) * finerSubdivideNum + idx];
    int nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
    OctNode rootNode = SubdivideNode[offset];
    int replacedId = rootNode.neighs[13];
    rootNode.neighs[13] = NodeArraySize + nowIdx;
    RebuildArray[nowIdx] = rootNode;

    ReplaceNodeId[nowIdx] = replacedId;

    RebuildDepthBuffer[nowIdx] = nowDepth;

    IsRoot[nowIdx] = 1;
    Point3D<float> thisNodeCenter;
    getNodeCenterAllDepth(rootNode.key, nowDepth, thisNodeCenter);
    RebuildCenterBuffer[nowIdx] = thisNodeCenter;

    int sonKey = (rootNode.key >> (3 * (device::maxDepth - nowDepth))) & 7;
    NodeArray[rootNode.parent].children[sonKey] = NodeArraySize + nowIdx;
    int parentNodeIdx;
    int childrenNums = 8;
    while (nowDepth < device::maxDepth) {
        nowDepth++;
        fixedDepthOffset = fixedDepthAddress[(nowDepth - 1) * finerSubdivideNum + idx];
        nowIdx = depthNodeAddress[nowDepth] + fixedDepthOffset;
        for (int j = 0; j < childrenNums; j += 8) {
            int fatherFixedDepthOffset = fixedDepthAddress[(nowDepth - 2) * finerSubdivideNum + idx];
            parentNodeIdx = depthNodeAddress[nowDepth - 1] + fatherFixedDepthOffset + j / 8;
            int parentGlobalIdx = RebuildArray[parentNodeIdx].neighs[13];
            int parentKey = RebuildArray[parentNodeIdx].key;
            for (int k = 0; k < 8; k++) {
                int thisRoundIdx = nowIdx + j + k;
                int nowKey = parentKey | (k << (3 * (device::maxDepth - nowDepth)));
                RebuildArray[thisRoundIdx].parent = parentGlobalIdx;
                RebuildArray[thisRoundIdx].key = nowKey;
                RebuildArray[thisRoundIdx].neighs[13] = NodeArraySize + thisRoundIdx;

                ReplaceNodeId[thisRoundIdx] = replacedId;

                RebuildDepthBuffer[thisRoundIdx] = nowDepth;

                getNodeCenterAllDepth(nowKey, nowDepth, thisNodeCenter);
                RebuildCenterBuffer[thisRoundIdx] = thisNodeCenter;

                RebuildArray[parentNodeIdx].children[k] = NodeArraySize + thisRoundIdx;
            }
        }
        childrenNums <<= 3;
    }
}

__global__ void SparseSurfelFusion::device::markValidMeshVertexIndex(const Point3D<float>* VertexBuffer, const unsigned int verticesNum, bool* markValidVertices)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= verticesNum) return;
    if (fabsf(VertexBuffer[idx].coords[0]) < device::eps) {
        printf("�������㷢������VertexBuffer[%d] = (%.5f, %.5f, %.5f)", idx, VertexBuffer[idx].coords[0], VertexBuffer[idx].coords[1], VertexBuffer[idx].coords[2]);
        markValidVertices[idx] = false;
    }
    else {
        markValidVertices[idx] = true;
    }
}

__global__ void SparseSurfelFusion::device::markValidMeshTriangleIndex(TriangleIndex* TriangleBuffer, const unsigned int previousVertexOffset, const unsigned int allTriNums, const unsigned int verticesNum, bool* markValidTriangleIndex)
{
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= allTriNums) return;
    bool triValid = true;
    TriangleIndex tri;
    for (int i = 0; i < 3; i++) {
        tri.idx[i] = TriangleBuffer[idx].idx[i] + previousVertexOffset;
        if (tri.idx[i] < 0 || tri.idx[i] >= verticesNum + previousVertexOffset) {
            triValid = false;
        }
    }

    TriangleBuffer[idx] = tri;
    if (triValid) {
        markValidTriangleIndex[idx] = true;
    }
    else {
        printf("���������������� index = %d   PreOffset = %d   verticesNum = %d   TriangleBuffer = (%d, %d, %d)\n", idx, previousVertexOffset, verticesNum, tri.idx[0], tri.idx[1], tri.idx[2]);
        markValidTriangleIndex[idx] = false;
    }
}



void SparseSurfelFusion::ComputeTriangleIndices::ComputeVertexImplicitFunctionValue(DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    const unsigned int VertexArraySize = VertexArray.Size();
    dim3 block(128);
    dim3 grid(divUp(VertexArraySize, block.x));

    device::ComputeVertexImplicitFunctionValueKernel << <grid, block, 0, stream >> > (VertexArray, NodeArray, BaseFunction, dx, encodeNodeIndexInFunction, VertexArraySize, isoValue, vvalue.Array().ptr());
}

void SparseSurfelFusion::ComputeTriangleIndices::insertTriangle(const Point3D<float>* VertexBufferHost, const int& allVexNums, const int* TriangleBufferHost, const int& allTriNums, CoredVectorMeshData& mesh)
{
    int previousVertex = mesh.inCorePoints.size();

    for (int i = 0; i < allVexNums; i++) {
        if (abs(VertexBufferHost[i].coords[0]) < EPSILON) {
            printf("error\n");
        }
        mesh.inCorePoints.push_back(VertexBufferHost[i]);
    }

    int inCoreFlag = 0; // �ж��ǵڼ����ڵ㣬�����λ�����˳��
    for (int i = 0; i < 3; i++) {
        inCoreFlag |= CoredMeshData::IN_CORE_FLAG[i];
    }

    for (int i = 0; i < allTriNums; i++) {
        TriangleIndex tri;
        for (int j = 0; j < 3; j++) {
            tri.idx[j] = TriangleBufferHost[3 * i + j] + previousVertex;
            //if (i % 100 == 0) printf("depth = %d   idx = %d   TriangleBufferHost[%d] = %d\n" ,depth, i, 3 * i + j, TriangleBufferHost[3 * i + j]);
            if (tri.idx[j] < 0 || tri.idx[j] >= allVexNums + previousVertex) {
                printf("%d %d\n", tri.idx[j] - previousVertex, allVexNums);
                printf("tri error\n");
            }
        }
        mesh.addTriangle(tri, inCoreFlag);
    }
}

void SparseSurfelFusion::ComputeTriangleIndices::insertTriangle(Point3D<float>* VertexBuffer, const int allVexNums, TriangleIndex* TriangleBuffer, const int allTriNums, hipStream_t stream)
{
    dim3 block_vex(128);
    dim3 grid_vex(divUp(allVexNums, block_vex.x));
    device::markValidMeshVertexIndex << <grid_vex, block_vex, 0, stream >> > (VertexBuffer, allVexNums, markValidTriangleVertex.Ptr());

    unsigned int* validVerticesCount = NULL;    // ��Ч�Ķ���
    unsigned int validVerticesCountHost = 0;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVerticesCount), sizeof(unsigned int), stream));

    void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
    size_t temp_storage_bytes_1 = 0;  // �м����
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, VertexBuffer, markValidTriangleVertex.Ptr(), MeshTriangleVertex.Ptr() + MeshTriangleVertex.ArraySize(), validVerticesCount, allVexNums, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, VertexBuffer, markValidTriangleVertex.Ptr(), MeshTriangleVertex.Ptr() + MeshTriangleVertex.ArraySize(), validVerticesCount, allVexNums, stream, false));	// ɸѡ	
    CHECKCUDA(hipMemcpyAsync(&validVerticesCountHost, validVerticesCount, sizeof(unsigned int), hipMemcpyDeviceToHost, stream));

    dim3 block_tri(128);
    dim3 grid_tri(divUp(allTriNums, block_tri.x));
    device::markValidMeshTriangleIndex << <grid_tri, block_tri, 0, stream >> > (TriangleBuffer, MeshTriangleVertex.ArraySize(), allTriNums, allVexNums, markValidTriangleIndex.Ptr());

    unsigned int* validTriangleIndicesCount = NULL;    // ��Ч������������������
    unsigned int validTriangleIndicesCountHost = 0;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validTriangleIndicesCount), sizeof(unsigned int), stream));

    void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
    size_t temp_storage_bytes_2 = 0;  // �м����
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, TriangleBuffer, markValidTriangleIndex.Ptr(), MeshTriangleIndex.Ptr() + MeshTriangleIndex.ArraySize(), validTriangleIndicesCount, allTriNums, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, TriangleBuffer, markValidTriangleIndex.Ptr(), MeshTriangleIndex.Ptr() + MeshTriangleIndex.ArraySize(), validTriangleIndicesCount, allTriNums, stream, false));	// ɸѡ	
    CHECKCUDA(hipMemcpyAsync(&validTriangleIndicesCountHost, validTriangleIndicesCount, sizeof(unsigned int), hipMemcpyDeviceToHost, stream));

    CHECKCUDA(hipStreamSynchronize(stream));
    MeshTriangleVertex.ResizeArrayOrException(validVerticesCountHost + MeshTriangleVertex.ArraySize());
    MeshTriangleIndex.ResizeArrayOrException(validTriangleIndicesCountHost + MeshTriangleIndex.ArraySize());
}

void SparseSurfelFusion::ComputeTriangleIndices::generateSubdivideNodeArrayCountAndAddress(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, const unsigned int OtherDepthNodeCount, hipStream_t stream)
{
    SubdivideNode.ResizeArrayOrException(OtherDepthNodeCount);
    CHECKCUDA(hipMemsetAsync(SubdivideNode.Array().ptr(), 0, sizeof(OctNode) * OtherDepthNodeCount, stream));

    int* SubdivideNodeNum = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideNodeNum), sizeof(int), stream));

    ////// �����޷�ʹ��hipcub::DeviceSelect::Flagged������API�ᵼ�¹����ڴ��������Ҫ������L1 Cache �� Share Memory�ı���
    //int* SubdivideNodeNumPtr = NULL;
    //CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideNodeNumPtr), sizeof(int), stream));
    //void* d_temp_storage = NULL;
    //size_t temp_storage_bytes = 0; 
    //CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr(), SubdivideNode.Array().ptr(), SubdivideNodeNumPtr, OtherDepthNodeCount, stream, false));	// ȷ����ʱ�豸�洢����
    //CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
    //CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr(), SubdivideNode.Array().ptr(), SubdivideNodeNumPtr, OtherDepthNodeCount, stream, false));	// ɸѡ
    //CHECKCUDA(hipMemcpyAsync(&SubdivideNodeNumHost, SubdivideNodeNumPtr, sizeof(int), hipMemcpyDeviceToHost, stream));
    //CHECKCUDA(hipStreamSynchronize(stream));

    // thrust::cuda::par.on(stream) -> ڹ��Thrust��ִ�в�������������ʽ
    thrust::device_ptr<OctNode> NodeArray_ptr = thrust::device_pointer_cast<OctNode>(NodeArray.Array().ptr());
    thrust::device_ptr<OctNode> SubdivideNode_ptr = thrust::device_pointer_cast<OctNode>(SubdivideNode.Array().ptr());
    thrust::device_ptr<OctNode> SubdivideNode_end = thrust::copy_if(thrust::cuda::par.on(stream), NodeArray_ptr, NodeArray_ptr + OtherDepthNodeCount, SubdivideNode_ptr, ifSubdivide());
    CHECKCUDA(hipStreamSynchronize(stream));
    SubdivideNodeNumHost = SubdivideNode_end - SubdivideNode_ptr;

    int* SubdivideDepthNum = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideDepthNum), sizeof(int) * SubdivideNodeNumHost * (Constants::maxDepth_Host + 1), stream));
    CHECKCUDA(hipMemsetAsync(SubdivideDepthNum, 0, sizeof(int) * SubdivideNodeNumHost * (Constants::maxDepth_Host + 1), stream));

    SubdivideDepthBuffer.ResizeArrayOrException(SubdivideNodeNumHost);
    
    dim3 block(128);
    dim3 grid(divUp(SubdivideNodeNumHost, block.x));
    device::precomputeSubdivideDepth << <grid, block, 0, stream >> > (SubdivideNode.ArrayView(), DepthBuffer, SubdivideNodeNumHost, SubdivideDepthBuffer.DeviceArray().ptr(), SubdivideDepthNum);

    int* subdivideDepthCount = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&subdivideDepthCount), sizeof(int), stream));


    for (int i = 0; i <= Constants::maxDepth_Host; i++) {
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, SubdivideDepthNum + i * SubdivideNodeNumHost, subdivideDepthCount, SubdivideNodeNumHost, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, SubdivideDepthNum + i * SubdivideNodeNumHost, subdivideDepthCount, SubdivideNodeNumHost, stream);
        CHECKCUDA(hipMemcpyAsync(&(SubdivideDepthCount[i]), subdivideDepthCount, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
    }

    CHECKCUDA(hipFreeAsync(SubdivideDepthNum, stream));    // ���꼴���ͷţ�����̫��
    CHECKCUDA(hipStreamSynchronize(stream));

    for (int i = 0; i <= Constants::maxDepth_Host; i++) {
        //printf("�� %d ��ϸ�ֽڵ�������%d   ", i, SubdivideDepthCount[i]);
        if (i == 0) SubdivideDepthAddress[i] = 0;
        else SubdivideDepthAddress[i] = SubdivideDepthAddress[i - 1] + SubdivideDepthCount[i - 1];
        //printf("�ڵ�ƫ�ƣ�%d\n", SubdivideDepthAddress[i]);
    }

}



void SparseSurfelFusion::ComputeTriangleIndices::generateVertexNumsAndVertexAddress(DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, hipStream_t stream)
{
    const unsigned int EdgeArraySize = EdgeArray.Size();
    vexNums.ResizeArrayOrException(EdgeArraySize);
    vexAddress.ResizeArrayOrException(EdgeArraySize);
    markValidVertex.ResizeArrayOrException(EdgeArraySize);
    dim3 block(128);
    dim3 grid(divUp(EdgeArraySize, block.x));
    device::generateVertexNumsKernel << <grid, block, 0, stream >> > (EdgeArray, NodeArray, vvalue, EdgeArraySize, vexNums.Array().ptr(), markValidVertex.Array().ptr());

    void* tempStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
    size_t tempStorageBytes = 0;
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, vexNums.Array().ptr(), vexAddress.Array().ptr(), EdgeArraySize, stream);
    CHECKCUDA(hipMallocAsync(&tempStorage, tempStorageBytes, stream));
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, vexNums.Array().ptr(), vexAddress.Array().ptr(), EdgeArraySize, stream);

    CHECKCUDA(hipFreeAsync(tempStorage, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::generateTriangleNumsAndTriangleAddress(DeviceArrayView<OctNode> NodeArray, DeviceArrayView<float> vvalue, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, hipStream_t stream)
{
    triNums.ResizeArrayOrException(DLevelNodeCount);
    cubeCatagory.ResizeArrayOrException(DLevelNodeCount);
    triAddress.ResizeArrayOrException(DLevelNodeCount);
    dim3 block(128);
    dim3 grid(divUp(DLevelNodeCount, block.x));
    device::generateTriangleNumsKernel << <grid, block, 0, stream >> > (NodeArray, vvalue, DLevelOffset, DLevelNodeCount, triNums.Array().ptr(), cubeCatagory.Array().ptr());

    void* tempStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
    size_t tempStorageBytes = 0;
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, triNums.Array().ptr(), triAddress.Array().ptr(), DLevelNodeCount, stream);
    CHECKCUDA(hipMallocAsync(&tempStorage, tempStorageBytes, stream));
    hipcub::DeviceScan::ExclusiveSum(tempStorage, tempStorageBytes, triNums.Array().ptr(), triAddress.Array().ptr(), DLevelNodeCount, stream);

    CHECKCUDA(hipFreeAsync(tempStorage, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::generateVerticesAndTriangle(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<VertexNode> VertexArray, DeviceArrayView<EdgeNode> EdgeArray, DeviceArrayView<FaceNode> FaceArray, const unsigned int DLevelOffset, const unsigned int DLevelNodeCount, hipStream_t stream)
{
    const unsigned int EdgeArraySize = EdgeArray.Size();
    const unsigned int FaceArraySize = FaceArray.Size();

    int lastVexAddr;
    int lastVexNums;
    CHECKCUDA(hipMemcpyAsync(&lastVexAddr, vexAddress.Array().ptr() + EdgeArraySize - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipMemcpyAsync(&lastVexNums, vexNums.Array().ptr() + EdgeArraySize - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������

    int allVexNums = lastVexAddr + lastVexNums; // vertex������

    Point3D<float>* VertexBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&VertexBuffer), sizeof(Point3D<float>) * allVexNums, stream));

    EdgeNode* validEdgeArray = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validEdgeArray), sizeof(EdgeNode) * allVexNums, stream));

    int* validVertexAddress = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVertexAddress), sizeof(int) * allVexNums, stream));

    int* validEdgeArrayNum = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validEdgeArrayNum), sizeof(int), stream));

    int* validVertexAddressNum = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&validVertexAddressNum), sizeof(int), stream));

    void* d_temp_storage_1 = NULL;
    size_t temp_storage_bytes_1 = 0;
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, EdgeArray.RawPtr(), markValidVertex.Array().ptr(), validEdgeArray, validEdgeArrayNum, EdgeArraySize, stream, false));	// ȷ����ʱ�豸�洢����
    CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, EdgeArray.RawPtr(), markValidVertex.Array().ptr(), validEdgeArray, validEdgeArrayNum, EdgeArraySize, stream, false));	// ɸѡ	

    void* d_temp_storage_2 = NULL;
    size_t temp_storage_bytes_2 = 0;
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, vexAddress.Array().ptr(), markValidVertex.Array().ptr(), validVertexAddress, validVertexAddressNum, EdgeArraySize, stream, false));
    CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
    CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, vexAddress.Array().ptr(), markValidVertex.Array().ptr(), validVertexAddress, validVertexAddressNum, EdgeArraySize, stream, false));

    assert(allVexNums == validEdgeArrayNum);        // �����Լ��
    assert(allVexNums == validVertexAddressNum);    // �����Լ��

    dim3 block_1(128);
    dim3 grid_1(divUp(allVexNums, block_1.x));
    device::generateIntersectionPoint << <grid_1, block_1, 0, stream >> > (NodeArray.ArrayView(), VertexArray, vvalue.ArrayView(), validEdgeArray, validVertexAddress, allVexNums, VertexBuffer);

    int lastTriAddr;
    int lastTriNums;
    CHECKCUDA(hipMemcpyAsync(&lastTriAddr, triAddress.Array().ptr() + DLevelNodeCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipMemcpyAsync(&lastTriNums, triNums.Array().ptr() + DLevelNodeCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������
    int allTriNums = lastTriAddr + lastTriNums;

    TriangleIndex* TriangleBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&TriangleBuffer), sizeof(TriangleIndex) * allTriNums, stream));

    int* hasSurfaceIntersection = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&hasSurfaceIntersection), sizeof(int) * FaceArraySize, stream));
    CHECKCUDA(hipMemsetAsync(hasSurfaceIntersection, 0, sizeof(int) * FaceArraySize, stream));

    dim3 block_2(128);
    dim3 grid_2(divUp(DLevelNodeCount, block_2.x));
    device::generateTrianglePos << <grid_2, block_2, 0, stream >> > (NodeArray.ArrayView(), FaceArray, triNums.ArrayView(), cubeCatagory.ArrayView(), vexAddress.ArrayView(), triAddress.ArrayView(), DLevelOffset, DLevelNodeCount, TriangleBuffer, hasSurfaceIntersection);
    CHECKCUDA(hipStreamSynchronize(stream));   // ������Ҫͳ��һ�����ж��������

    insertTriangle(VertexBuffer, allVexNums, TriangleBuffer, allTriNums, stream);

    markValidSubdividedNode.ResizeArrayOrException(DLevelOffset);

    dim3 block_3(128);
    dim3 grid_3(divUp(DLevelOffset, block_3.x));
    device::ProcessLeafNodesAtOtherDepth << <grid_3, block_3, 0, stream >> > (VertexArray, vvalue.ArrayView(), DLevelOffset, hasSurfaceIntersection, NodeArray.Array().ptr(), markValidSubdividedNode.Array().ptr());

    // �������꼴�ͷ�
    CHECKCUDA(hipFreeAsync(VertexBuffer, stream));
    CHECKCUDA(hipFreeAsync(validEdgeArray, stream));
    CHECKCUDA(hipFreeAsync(validVertexAddress, stream));
    CHECKCUDA(hipFreeAsync(validEdgeArrayNum, stream));
    CHECKCUDA(hipFreeAsync(validVertexAddressNum, stream));
    CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));
    CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));
    CHECKCUDA(hipFreeAsync(TriangleBuffer, stream));
    CHECKCUDA(hipFreeAsync(hasSurfaceIntersection, stream));

}


void SparseSurfelFusion::ComputeTriangleIndices::CoarserSubdivideNodeAndRebuildMesh(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, DeviceArrayView<Point3D<float>> CenterBuffer, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    int minSubdivideRootDepth;
    SubdivideDepthBuffer.SynchronizeToHost(stream);
    std::vector<int>& SubdivideDepthBufferHost = SubdivideDepthBuffer.HostArray();
    std::vector<OctNode> SubdivideNodeHost;
    SubdivideNode.ArrayView().Download(SubdivideNodeHost);
    minSubdivideRootDepth = SubdivideDepthBufferHost[0];

    int maxNodeNums = (powf(8, (Constants::maxDepth_Host - minSubdivideRootDepth + 1)) - 1) / 7;

    EasyOctNode* SubdivideArray = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArray), sizeof(EasyOctNode) * maxNodeNums, stream));

    int* SubdivideArrayDepthBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArrayDepthBuffer), sizeof(int) * maxNodeNums, stream));

    Point3D<float>* SubdivideArrayCenterBuffer = NULL;
    CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideArrayCenterBuffer), sizeof(Point3D<float>) * maxNodeNums, stream));

    for (int i = 0; i < SubdivideNodeNumHost; i++) {
        int rootDepth = SubdivideDepthBufferHost[i];
        if (rootDepth >= finerDepth) break;

        int SubdivideArraySize = (powf(8, (Constants::maxDepth_Host - rootDepth + 1)) - 1) / 7;
        int currentNodeNum = 1;
        for (int j = rootDepth; j <= Constants::maxDepth_Host; j++) {
            fixedDepthNodeNum[j] = currentNodeNum;
            currentNodeNum <<= 3;       // ��8
        }

        //for (int j = 0; j <= Constants::maxDepth_Host; j++) {
        //    printf("fixedDepthNodeNum[%d] = %d\n", j, fixedDepthNodeNum[j]);
        //}

        //printf("SubdivideArraySize - (D-1) = %d   fixedDepthNodeNum = %d\n", SubdivideArraySize - fixedDepthNodeNum[Constants::maxDepth_Host - 1], fixedDepthNodeNum[Constants::maxDepth_Host]);

        for (int j = rootDepth; j <= Constants::maxDepth_Host; j++) {
            fixedDepthNodeAddress[j] = fixedDepthNodeAddress[j - 1] + fixedDepthNodeNum[j - 1];
        }
        OctNode rootNode = SubdivideNodeHost[i];
        int rootIndex = rootNode.neighs[13];
        int rootParent = rootNode.parent;
        int rootKey = rootNode.key;
        int rootSonKey = (rootKey >> (3 * (Constants::maxDepth_Host - rootDepth))) & 7;

        CHECKCUDA(hipMemsetAsync(SubdivideArray, 0, sizeof(EasyOctNode) * SubdivideArraySize, stream));

        int NodeArraySize = NodeArray.ArraySize();
        OctNode* NodeArrayPtr = NodeArray.Array().ptr();
        CHECKCUDA(hipMemcpyAsync(&NodeArrayPtr[rootParent].children[rootSonKey], &NodeArraySize, sizeof(int), hipMemcpyHostToDevice, stream));
        CHECKCUDA(hipMemcpyAsync(&SubdivideArray[0].parent, &rootParent, sizeof(int), hipMemcpyHostToDevice, stream));

        dim3 block_1(128);
        dim3 grid_1(divUp(SubdivideArraySize, block_1.x));
        device::singleRebuildArray << <grid_1, block_1, 0, stream >> > (SubdivideNode.ArrayView(), SubdivideDepthBuffer.DeviceArrayReadOnly(), i, NodeArraySize, SubdivideArraySize, SubdivideArray, SubdivideArrayDepthBuffer, SubdivideArrayCenterBuffer);

        for (int depth = rootDepth; depth <= Constants::maxDepth_Host; depth++) {
            dim3 block_2(128);
            dim3 grid_2(divUp(fixedDepthNodeNum[depth], block_2.x));
            device::computeRebuildNeighbor << <grid_2, block_2, 0, stream >> > (NodeArray.ArrayView(), fixedDepthNodeAddress[depth], fixedDepthNodeNum[depth], NodeArraySize, depth, SubdivideArray);
        }

        /**************************************** SubdivideVertexArray ****************************************/

        VertexNode* SubdividePreVertexArray = NULL;     // ��ʱ���������꼴ɾ
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdividePreVertexArray), sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdividePreVertexArray, 0, sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        markValidSubdivideVertex.ResizeArrayOrException(fixedDepthNodeNum[Constants::maxDepth_Host]);

        dim3 block_3(128);
        dim3 grid_3(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_3.x));
        device::initSubdivideVertexOwner << <grid_3, block_3, 0, stream >> > (SubdivideArray, SubdivideArrayCenterBuffer, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], NodeArraySize, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr());
    
        VertexNode* SubdivideVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexArray), sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVertexArray, 0, sizeof(VertexNode) * 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
    
        int* SubdivideVertexArraySize = NULL;
        int SubdivideVertexArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexArraySize), sizeof(int), stream));

        void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_1 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr(), SubdivideVertexArray, SubdivideVertexArraySize, 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, SubdividePreVertexArray, markValidSubdivideVertex.Array().ptr(), SubdivideVertexArray, SubdivideVertexArraySize, 8 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideVertexArraySizeHost, SubdivideVertexArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdividePreVertexArray, stream));  // ��ʱ���������꼴ɾ
        CHECKCUDA(hipFreeAsync(SubdivideVertexArraySize, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));

        CHECKCUDA(hipStreamSynchronize(stream));       // ͬ���������SubdivideVertexArraySizeHost

        //printf("SubdivideVertexArraySizeHost = %d\n", SubdivideVertexArraySizeHost);


        dim3 block_4(128);
        dim3 grid_4(divUp(SubdivideVertexArraySizeHost, block_4.x));
        device::maintainSubdivideVertexNodePointer << <grid_4, block_4, 0, stream >> > (CenterBuffer, SubdivideVertexArraySizeHost, NodeArraySize, SubdivideArrayCenterBuffer, SubdivideVertexArray, SubdivideArray);

        /**************************************** SubdivideEdgeArray ****************************************/

        EdgeNode* SubdividePreEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdividePreEdgeArray), sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdividePreEdgeArray, 0, sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        markValidSubdivideEdge.ResizeArrayOrException(fixedDepthNodeNum[Constants::maxDepth_Host]);

        dim3 block_5(128);
        dim3 grid_5(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_5.x));
        device::initSubdivideEdgeArray << <grid_5, block_5, 0, stream >> > (SubdivideArray, SubdivideArrayCenterBuffer, NodeArraySize, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr());

        EdgeNode* SubdivideEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideEdgeArray), sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideEdgeArray, 0, sizeof(EdgeNode) * 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        int* SubdivideEdgeArraySize = NULL;
        int SubdivideEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_2 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr(), SubdivideEdgeArray, SubdivideEdgeArraySize, 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, SubdividePreEdgeArray, markValidSubdivideEdge.Array().ptr(), SubdivideEdgeArray, SubdivideEdgeArraySize, 12 * fixedDepthNodeNum[Constants::maxDepth_Host], stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideEdgeArraySizeHost, SubdivideEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdividePreEdgeArray, stream));  // ��ʱ���������꼴ɾ
        CHECKCUDA(hipFreeAsync(SubdivideEdgeArraySize, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));

        CHECKCUDA(hipStreamSynchronize(stream));       // ͬ���������SubdivideEdgeArraySizeHost

        //printf("SubdivideEdgeArraySizeHost = %d\n", SubdivideEdgeArraySizeHost);


        dim3 block_6(128);
        dim3 grid_6(divUp(SubdivideEdgeArraySizeHost, block_6.x));
        device::maintainSubdivideEdgeNodePointer << <grid_6, block_6, 0, stream >> > (CenterBuffer, SubdivideArrayCenterBuffer, SubdivideEdgeArraySizeHost, NodeArraySize, SubdivideArray, SubdivideEdgeArray);

        /**************************************** ����ϸ�ֽڵ���ʽ������ֵ, ����ϸ�ֶ����vexNums��vexAddress ****************************************/
        float* SubdivideVvalue = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVvalue), sizeof(float) * SubdivideVertexArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVvalue, 0, sizeof(float) * SubdivideVertexArraySizeHost, stream));
        dim3 block_7(128);
        dim3 grid_7(divUp(SubdivideVertexArraySizeHost, block_7.x));
        device::computeSubdivideVertexImplicitFunctionValue << <grid_7, block_7, 0, stream >> > (SubdivideVertexArray, SubdivideArray, NodeArray.ArrayView(), dx, encodeNodeIndexInFunction, BaseFunction, NodeArraySize, rootIndex, SubdivideVertexArraySizeHost, isoValue, SubdivideVvalue);

        int* SubdivideVexNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVexNums), sizeof(int) * SubdivideEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVexNums, 0, sizeof(int) * SubdivideEdgeArraySizeHost, stream));

        markValidSubdivedeVexNum.ResizeArrayOrException(SubdivideEdgeArraySizeHost);
        
        dim3 block_8(128);
        dim3 grid_8(divUp(SubdivideEdgeArraySizeHost, block_8.x));
        device::generateSubdivideVexNums << <grid_8, block_8, 0, stream >> > (SubdivideEdgeArray, SubdivideArray, SubdivideEdgeArraySizeHost, NodeArraySize, SubdivideVvalue, SubdivideVexNums, markValidSubdivedeVexNum.Array().ptr());
        
        
        //printf("SubdivideEdgeArraySizeHost = %d\n", SubdivideEdgeArraySizeHost);


        int* SubdivideVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVexAddress), sizeof(int) * SubdivideEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(SubdivideVexAddress, 0, sizeof(int) * SubdivideEdgeArraySizeHost, stream));

        void* tempVexAddressStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
        size_t tempVexAddressStorageBytes = 0;
        hipcub::DeviceScan::ExclusiveSum(tempVexAddressStorage, tempVexAddressStorageBytes, SubdivideVexNums, SubdivideVexAddress, SubdivideEdgeArraySizeHost, stream);
        CHECKCUDA(hipMallocAsync(&tempVexAddressStorage, tempVexAddressStorageBytes, stream));
        hipcub::DeviceScan::ExclusiveSum(tempVexAddressStorage, tempVexAddressStorageBytes, SubdivideVexNums, SubdivideVexAddress, SubdivideEdgeArraySizeHost, stream);

        CHECKCUDA(hipFreeAsync(tempVexAddressStorage, stream));

        int SubdivideLastVexAddr = -1;
        int SubdivideLastVexNums = -1;
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastVexAddr, SubdivideVexAddress + SubdivideEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastVexNums, SubdivideVexNums + SubdivideEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));


        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��

        int SubdivideAllVexNums = SubdivideLastVexAddr + SubdivideLastVexNums;
        //printf("SubdivideAllVexNums = %d\n", SubdivideAllVexNums);

        if (SubdivideAllVexNums == 0) {
            CHECKCUDA(hipMemcpyAsync(&(NodeArray[rootParent].children[rootSonKey]), &rootIndex, sizeof(int), hipMemcpyHostToDevice, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
            CHECKCUDA(hipFreeAsync(SubdivideEdgeArray, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVvalue, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVexNums, stream));
            CHECKCUDA(hipFreeAsync(SubdivideVexAddress, stream));
            continue;
        }

        /**************************************** ����ϸ�ֶ���������κ����������� ****************************************/

        int* SubdivideTriNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriNums), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideTriNums, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        int* SubdivideCubeCatagory = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideCubeCatagory), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideCubeCatagory, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        dim3 block_9(128);
        dim3 grid_9(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_9.x));
        device::generateTriNums << <grid_9, block_9, 0, stream >> > (SubdivideArray, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdivideVvalue, SubdivideTriNums, SubdivideCubeCatagory);

        int* SubdivideTriAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriAddress), sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));
        CHECKCUDA(hipMemsetAsync(SubdivideTriAddress, 0, sizeof(int) * fixedDepthNodeNum[Constants::maxDepth_Host], stream));

        void* tempTriAddressStorage = NULL;	//���㷨��ʱ���������꼴�ͷš�����ǰ׺�͵���ʱ����
        size_t tempTriAddressStorageBytes = 0;
        hipcub::DeviceScan::ExclusiveSum(tempTriAddressStorage, tempTriAddressStorageBytes, SubdivideTriNums, SubdivideTriAddress, fixedDepthNodeNum[Constants::maxDepth_Host], stream);
        CHECKCUDA(hipMallocAsync(&tempTriAddressStorage, tempTriAddressStorageBytes, stream));
        hipcub::DeviceScan::ExclusiveSum(tempTriAddressStorage, tempTriAddressStorageBytes, SubdivideTriNums, SubdivideTriAddress, fixedDepthNodeNum[Constants::maxDepth_Host], stream);

        CHECKCUDA(hipFreeAsync(tempTriAddressStorage, stream));

        Point3D<float>* SubdivideVertexBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideVertexBuffer), sizeof(Point3D<float>) * SubdivideAllVexNums, stream));
        //std::vector<Point3D<float>> SubdivideVertexBufferHost;
        //SubdivideVertexBufferHost.resize(SubdivideAllVexNums);

        EdgeNode* SubdivideValidEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidEdgeArray), sizeof(EdgeNode) * SubdivideAllVexNums, stream));

        int* SubdivideValidEdgeArraySize = NULL;    // ��Ч��ϸ�ֱ�device
        int SubdivideValidEdgeArraySizeHost = -1;   // ��Ч��ϸ�ֱ�Host
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_3 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_3 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_3, temp_storage_bytes_3, SubdivideEdgeArray, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidEdgeArray, SubdivideValidEdgeArraySize, SubdivideEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_3, temp_storage_bytes_3, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_3, temp_storage_bytes_3, SubdivideEdgeArray, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidEdgeArray, SubdivideValidEdgeArraySize, SubdivideEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideValidEdgeArraySizeHost, SubdivideValidEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        int* SubdivideValidVexAddress = NULL;    // ��Ч��ϸ�ֱ�device
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidVexAddress), sizeof(int) * SubdivideAllVexNums, stream));

        int* SubdivideValidVexAddressSize = NULL;    // ��Ч��ϸ�ֱ�device
        int SubdivideValidVexAddressSizeHost = -1;   // ��Ч��ϸ�ֱ�Host
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideValidVexAddressSize), sizeof(int), stream));

        void* d_temp_storage_4 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_4 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_4, temp_storage_bytes_4, SubdivideVexAddress, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidVexAddress, SubdivideValidVexAddressSize, SubdivideEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_4, temp_storage_bytes_4, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_4, temp_storage_bytes_4, SubdivideVexAddress, markValidSubdivedeVexNum.Array().ptr(), SubdivideValidVexAddress, SubdivideValidVexAddressSize, SubdivideEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&SubdivideValidVexAddressSizeHost, SubdivideValidVexAddressSize, sizeof(int), hipMemcpyDeviceToHost, stream));

        dim3 block_10(128);
        dim3 grid_10(divUp(SubdivideAllVexNums, block_10.x));
        device::generateSubdivideIntersectionPoint << <grid_10, block_10, 0, stream >> > (SubdivideValidEdgeArray, SubdivideVertexArray, SubdivideArray, SubdivideValidVexAddress, SubdivideVvalue, SubdivideValidEdgeArraySizeHost, NodeArraySize, SubdivideVertexBuffer);
        //CHECKCUDA(hipMemcpyAsync(SubdivideVertexBufferHost.data(), SubdivideVertexBuffer, sizeof(Point3D<float>) * SubdivideAllVexNums, hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(SubdivideValidEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideValidVexAddress, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_3, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage_4, stream));

        //CHECKCUDA(hipStreamSynchronize(stream));
        //printf("depth = %d   SubdivideValidVexAddressSize = %d\n", i, SubdivideValidVexAddressSizeHost);

        int SubdivideLastTriAddr;
        int SubdivideLastTriNums;
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastTriAddr, SubdivideTriAddress + fixedDepthNodeNum[Constants::maxDepth_Host] - 1, sizeof(int), hipMemcpyDeviceToHost));
        CHECKCUDA(hipMemcpyAsync(&SubdivideLastTriNums, SubdivideTriNums + fixedDepthNodeNum[Constants::maxDepth_Host] - 1, sizeof(int), hipMemcpyDeviceToHost));
        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        int SubdivideAllTriNums = SubdivideLastTriAddr + SubdivideLastTriNums;
        //printf("depth = %d   SubdivideAllTriNums = %d\n", i, SubdivideAllTriNums);

        TriangleIndex* SubdivideTriangleBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&SubdivideTriangleBuffer), sizeof(TriangleIndex) * SubdivideAllTriNums, stream));
        //CHECKCUDA(hipMemsetAsync(SubdivideTriangleBuffer, 0, sizeof(TriangleIndex) * SubdivideAllTriNums, stream));

        //std::vector<int> SubdivideTriangleBufferHost;
        //SubdivideTriangleBufferHost.resize(3 * SubdivideAllTriNums);

        dim3 block_11(128);
        dim3 grid_11(divUp(fixedDepthNodeNum[Constants::maxDepth_Host], block_11.x));
        device::generateSubdivideTrianglePos << <grid_11, block_11, 0, stream >> > (SubdivideArray, fixedDepthNodeAddress[Constants::maxDepth_Host], fixedDepthNodeNum[Constants::maxDepth_Host], SubdivideTriNums, SubdivideCubeCatagory, SubdivideVexAddress, SubdivideTriAddress, SubdivideTriangleBuffer);



        insertTriangle(SubdivideVertexBuffer, SubdivideAllVexNums, SubdivideTriangleBuffer, SubdivideAllTriNums, stream);
        //CHECKCUDA(hipMemcpyAsync(SubdivideTriangleBufferHost.data(), SubdivideTriangleBuffer, sizeof(int) * 3 * SubdivideAllTriNums, hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        


        //insertTriangle(SubdivideVertexBufferHost.data(), SubdivideAllVexNums, SubdivideTriangleBufferHost.data(), SubdivideAllTriNums, mesh);

        CHECKCUDA(hipMemcpy(&(NodeArray[rootParent].children[rootSonKey]), &rootIndex, sizeof(int), hipMemcpyHostToDevice));
        CHECKCUDA(hipFreeAsync(SubdivideVertexArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVvalue, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVexNums, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVexAddress, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriNums, stream));
        CHECKCUDA(hipFreeAsync(SubdivideCubeCatagory, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriAddress, stream));
        CHECKCUDA(hipFreeAsync(SubdivideVertexBuffer, stream));
        CHECKCUDA(hipFreeAsync(SubdivideTriangleBuffer, stream));
    }
    CHECKCUDA(hipFreeAsync(SubdivideArray, stream));
    CHECKCUDA(hipFreeAsync(SubdivideArrayCenterBuffer, stream));
    CHECKCUDA(hipFreeAsync(SubdivideArrayDepthBuffer, stream));
}

void SparseSurfelFusion::ComputeTriangleIndices::FinerSubdivideNodeAndRebuildMesh(DeviceBufferArray<OctNode>& NodeArray, DeviceArrayView<unsigned int> DepthBuffer, DeviceArrayView<Point3D<float>> CenterBuffer, DeviceArrayView<ConfirmedPPolynomial<CONVTIMES + 1, CONVTIMES + 2>> BaseFunction, DeviceArrayView<float> dx, DeviceArrayView<int> encodeNodeIndexInFunction, const float isoValue, hipStream_t stream)
{
    const unsigned int NodeArraySize = NodeArray.ArraySize();
    for (int i = finerDepth; i < Constants::maxDepth_Host; i++) {
        int finerDepthStart = SubdivideDepthAddress[i];
        int finerSubdivideNum = SubdivideDepthCount[i];
        int* fixedDepthNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&fixedDepthNums), sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        CHECKCUDA(hipMemsetAsync(fixedDepthNums, 0, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));

        dim3 block_1(128);
        dim3 grid_1(divUp(finerSubdivideNum, block_1.x));
        device::initFixedDepthNums << <grid_1, block_1, 0, stream >> > (SubdivideNode.ArrayView(), SubdivideDepthBuffer.DeviceArrayReadOnly(), finerDepthStart, finerSubdivideNum, fixedDepthNums);

        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //std::vector<int> fixedDepthNumsHost;
        //fixedDepthNumsHost.resize(finerSubdivideNum * Constants::maxDepth_Host);
        //CHECKCUDA(hipMemcpyAsync(fixedDepthNumsHost.data(), fixedDepthNums, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, hipMemcpyDeviceToHost, stream));
        //for (int j = 0; j < fixedDepthNumsHost.size(); j++) {
        //    if (j % 1000 == 0) printf("depth = %d   fixedDepthNumsHost[%d] = %d\n", i, j, fixedDepthNumsHost[j]);
        //}        


        int* rebuildNumsDevice = NULL;
        int rebuildNums = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&rebuildNumsDevice), sizeof(int), stream));
        
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, fixedDepthNums, rebuildNumsDevice, finerSubdivideNum * Constants::maxDepth_Host, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, fixedDepthNums, rebuildNumsDevice, finerSubdivideNum * Constants::maxDepth_Host, stream);
        CHECKCUDA(hipMemcpyAsync(&rebuildNums, rebuildNumsDevice, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
        CHECKCUDA(hipFreeAsync(rebuildNumsDevice, stream));
        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //printf("depth = %d   rebuildNums = %d\n", i, rebuildNums);

        for (int depth = 1; depth <= Constants::maxDepth_Host; depth++) {
            void* d_temp_storage_1 = NULL;
            size_t temp_storage_bytes_1 = 0;
            int* LevelNodeCount = NULL;
            CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&LevelNodeCount), sizeof(int), stream));
            hipcub::DeviceReduce::Sum(d_temp_storage_1, temp_storage_bytes_1, fixedDepthNums + (depth - 1) * finerSubdivideNum, LevelNodeCount, finerSubdivideNum, stream);
            CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
            hipcub::DeviceReduce::Sum(d_temp_storage_1, temp_storage_bytes_1, fixedDepthNums + (depth - 1) * finerSubdivideNum, LevelNodeCount, finerSubdivideNum, stream);
            CHECKCUDA(hipMemcpyAsync(&depthNodeCount[depth], LevelNodeCount, sizeof(int), hipMemcpyDeviceToHost, stream));
            CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));
            CHECKCUDA(hipFreeAsync(LevelNodeCount, stream));
        }
        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //for (int depth = 0; depth <= Constants::maxDepth_Host; depth++) {
        //    printf("depth = %d   depthNodeCount[%d] = %d\n", i, depth, depthNodeCount[depth]);
        //}

        for (int depth = 0; depth <= Constants::maxDepth_Host; depth++) {
            if (depth == 0) depthNodeAddress[depth] = 0;
            else {
                depthNodeAddress[depth] = depthNodeAddress[depth - 1] + depthNodeCount[depth - 1];
            }
        }

        int* depthNodeAddress_Device = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&depthNodeAddress_Device), sizeof(int) * (Constants::maxDepth_Host + 1), stream));
        CHECKCUDA(hipMemcpyAsync(depthNodeAddress_Device, depthNodeAddress, sizeof(int) * (Constants::maxDepth_Host + 1), hipMemcpyHostToDevice, stream));

        int* fixedDepthAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&fixedDepthAddress), sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        CHECKCUDA(hipMemsetAsync(fixedDepthAddress, 0, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, stream));
        for (int depth = 1; depth <= Constants::maxDepth_Host; depth++) {
            void* d_temp_storage = NULL;
            size_t temp_storage_bytes = 0;
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, fixedDepthNums + (depth - 1) * finerSubdivideNum, fixedDepthAddress + (depth - 1) * finerSubdivideNum, finerSubdivideNum, stream);
            CHECKCUDA(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
            hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, fixedDepthNums + (depth - 1) * finerSubdivideNum, fixedDepthAddress + (depth - 1) * finerSubdivideNum, finerSubdivideNum, stream);
            CHECKCUDA(hipFreeAsync(d_temp_storage, stream));
        }

        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //std::vector<int> fixedDepthAddressTest;
        //fixedDepthAddressTest.resize(finerSubdivideNum * Constants::maxDepth_Host);
        //CHECKCUDA(hipMemcpy(fixedDepthAddressTest.data(), fixedDepthAddress, sizeof(int) * finerSubdivideNum * Constants::maxDepth_Host, hipMemcpyDeviceToHost));
        //for (int j = 0; j < fixedDepthAddressTest.size(); j++) {
        //    if (j % 1000 == 0) printf("depth = %d   fixedDepthAddress[%d] = %d\n", i, j, fixedDepthAddressTest[j]);
        //}

        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ�������rebuildNums

        const unsigned int rebuildDLevelCount = rebuildNums - depthNodeAddress[Constants::maxDepth_Host];
        //printf("Depth = %d  rebuildNums = %d  depthNodeCount[max] = %d\n", i, rebuildNums, depthNodeCount[Constants::maxDepth_Host]);

        EasyOctNode* RebuildArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildArray), sizeof(EasyOctNode) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildArray, 0, sizeof(EasyOctNode) * rebuildNums, stream));

        int* RebuildDepthBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildDepthBuffer), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildDepthBuffer, 0, sizeof(int) * rebuildNums, stream));

        Point3D<float>* RebuildCenterBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildCenterBuffer), sizeof(Point3D<float>) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(RebuildCenterBuffer, 0, sizeof(Point3D<float>) * rebuildNums, stream));

        int* ReplaceNodeId = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&ReplaceNodeId), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(ReplaceNodeId, 0, sizeof(int) * rebuildNums, stream));

        int* IsRoot = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&IsRoot), sizeof(int) * rebuildNums, stream));
        CHECKCUDA(hipMemsetAsync(IsRoot, 0, sizeof(int) * rebuildNums, stream));

        //printf("depth = %d   finerSubdivideNum = %d\n", i, finerSubdivideNum);
        //printf("depth = %d   finerDepthStart = %d\n", i, finerDepthStart);
        dim3 block_2(128);
        dim3 grid_2(divUp(finerSubdivideNum, block_2.x));
        device::wholeRebuildArray << <grid_2, block_2, 0, stream >> > (SubdivideNode.ArrayView(), finerDepthStart, finerSubdivideNum, NodeArraySize, SubdivideDepthBuffer.DeviceArray().ptr(), depthNodeAddress_Device, fixedDepthAddress, RebuildArray, RebuildDepthBuffer, RebuildCenterBuffer, ReplaceNodeId, IsRoot, NodeArray.Array().ptr());
        
        //CHECKCUDA(hipStreamSynchronize(stream));
        //std::vector<EasyOctNode> RebuildArrayTest;
        //RebuildArrayTest.resize(rebuildNums);
        //CHECKCUDA(hipMemcpyAsync(RebuildArrayTest.data(), RebuildArray, sizeof(EasyOctNode) * rebuildNums, hipMemcpyDeviceToHost, stream));
        //for (int k = 0; k < RebuildArrayTest.size(); k++) {
        //    if (i == finerDepth && k % 1000 == 0) printf("idx = %d    RebuildArray = %d\n", k, RebuildArrayTest[k].children[3]);
        //}

        for (int depth = finerDepth; depth <= Constants::maxDepth_Host; depth++) {
            dim3 block(128);
            dim3 grid(divUp(depthNodeCount[depth], block.x));
            device::computeRebuildNeighbor << <grid, block, 0, stream >> > (NodeArray.ArrayView(), depthNodeAddress[depth], depthNodeCount[depth], NodeArraySize, depth, RebuildArray);
        }

        //CHECKCUDA(hipStreamSynchronize(stream));
        //std::vector<EasyOctNode> RebuildArrayTest;
        //RebuildArrayTest.resize(rebuildNums);
        //CHECKCUDA(hipMemcpyAsync(RebuildArrayTest.data(), RebuildArray, sizeof(EasyOctNode) * rebuildNums, hipMemcpyDeviceToHost, stream));
        //for (int k = 0; k < RebuildArrayTest.size(); k++) {
        //    if (i == finerDepth && k % 1000 == 0) printf("idx = %d    RebuildArray = %d\n", k, RebuildArrayTest[k].neighs[8]);
        //}

        VertexNode* RebuildPreVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildPreVertexArray), sizeof(VertexNode) * rebuildDLevelCount * 8, stream));
        CHECKCUDA(hipMemsetAsync(RebuildPreVertexArray, 0, sizeof(VertexNode) * rebuildDLevelCount * 8, stream));

        markValidFinerVexArray.ResizeArrayOrException(rebuildDLevelCount * 8);

        dim3 block_3(128);
        dim3 grid_3(divUp(rebuildDLevelCount, block_3.x));
        device::initSubdivideVertexOwner << <grid_3, block_3, 0, stream >> > (RebuildArray, RebuildCenterBuffer, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, NodeArraySize, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr());
        
        VertexNode* RebuildVertexArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexArray), sizeof(VertexNode) * rebuildDLevelCount * 8, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVertexArray, 0, sizeof(VertexNode) * rebuildDLevelCount * 8, stream));

        int* RebuildVertexArraySize = NULL;
        int RebuildVertexArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexArraySize), sizeof(int), stream));

        void* d_temp_storage_1 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_1 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr(), RebuildVertexArray, RebuildVertexArraySize, 8 * rebuildDLevelCount, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_1, temp_storage_bytes_1, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_1, temp_storage_bytes_1, RebuildPreVertexArray, markValidFinerVexArray.Array().ptr(), RebuildVertexArray, RebuildVertexArraySize, 8 * rebuildDLevelCount, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildVertexArraySizeHost, RebuildVertexArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(RebuildPreVertexArray, stream));   // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(RebuildVertexArraySize, stream));  // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(d_temp_storage_1, stream));        // ��ʱ��������ʱɾ��

        //CHECKCUDA(hipStreamSynchronize(stream));
        //printf("depth = %d  RebuildVertexArraySize = %d\n", i, RebuildVertexArraySizeHost);

        dim3 block_4(128);
        dim3 grid_4(divUp(RebuildVertexArraySizeHost, block_4.x));
        device::maintainSubdivideVertexNodePointer << <grid_4, block_4, 0, stream >> > (CenterBuffer, RebuildVertexArraySizeHost, NodeArraySize, RebuildCenterBuffer, RebuildVertexArray, RebuildArray);

        EdgeNode* RebuildPreEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildPreEdgeArray), sizeof(EdgeNode) * rebuildDLevelCount * 12, stream));
        CHECKCUDA(hipMemsetAsync(RebuildPreEdgeArray, 0, sizeof(EdgeNode) * rebuildDLevelCount * 12, stream));

        markValidFinerEdge.ResizeArrayOrException(rebuildDLevelCount * 12);

        dim3 block_5(128);
        dim3 grid_5(divUp(rebuildDLevelCount, block_5.x));
        device::initSubdivideEdgeArray << <grid_5, block_5, 0, stream >> > (RebuildArray, RebuildCenterBuffer, NodeArraySize, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr());

        EdgeNode* RebuildEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildEdgeArray), sizeof(EdgeNode)* rebuildDLevelCount * 12, stream));
        CHECKCUDA(hipMemsetAsync(RebuildEdgeArray, 0, sizeof(EdgeNode)* rebuildDLevelCount * 12, stream));

        int* RebuildEdgeArraySize = NULL;
        int RebuildEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_2 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_2 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr(), RebuildEdgeArray, RebuildEdgeArraySize, 12 * rebuildDLevelCount, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_2, temp_storage_bytes_2, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_2, temp_storage_bytes_2, RebuildPreEdgeArray, markValidFinerEdge.Array().ptr(), RebuildEdgeArray, RebuildEdgeArraySize, 12 * rebuildDLevelCount, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildEdgeArraySizeHost, RebuildEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(RebuildPreEdgeArray, stream));      // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(RebuildEdgeArraySize, stream));     // ��ʱ��������ʱɾ��
        CHECKCUDA(hipFreeAsync(d_temp_storage_2, stream));         // ��ʱ��������ʱɾ��

        //CHECKCUDA(hipStreamSynchronize(stream));
        //printf("depth = %d  RebuildVertexArraySize = %d  RebuildEdgeArraySize = %d\n", i, RebuildVertexArraySizeHost, RebuildEdgeArraySizeHost);

        dim3 block_6(128);
        dim3 grid_6(divUp(RebuildEdgeArraySizeHost, block_6.x));
        device::maintainSubdivideEdgeNodePointer << <grid_6, block_6, 0, stream >> > (CenterBuffer, RebuildCenterBuffer, RebuildEdgeArraySizeHost, NodeArraySize, RebuildArray, RebuildEdgeArray);

        float* RebuildVvalue = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVvalue), sizeof(float) * RebuildVertexArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVvalue, 0, sizeof(float) * RebuildVertexArraySizeHost, stream));

        dim3 block_7(128);
        dim3 grid_7(divUp(RebuildVertexArraySizeHost, block_7.x));
        device::computeSubdivideVertexImplicitFunctionValue << <grid_7, block_7, 0, stream >> > (RebuildVertexArray, RebuildArray, NodeArray.ArrayView(), dx, encodeNodeIndexInFunction, BaseFunction, NodeArraySize, ReplaceNodeId, IsRoot, RebuildVertexArraySizeHost, isoValue, RebuildVvalue);

        CHECKCUDA(hipFreeAsync(ReplaceNodeId, stream));
        CHECKCUDA(hipFreeAsync(IsRoot, stream));

        int* RebuildVexNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVexNums), sizeof(int) * RebuildEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVexNums, 0, sizeof(int) * RebuildEdgeArraySizeHost, stream));

        dim3 block_8(128);
        dim3 grid_8(divUp(RebuildEdgeArraySizeHost, block_8.x));
        device::generateSubdivideVexNums << <grid_8, block_8, 0, stream >> > (RebuildEdgeArray, RebuildArray, RebuildEdgeArraySizeHost, NodeArraySize, RebuildVvalue, RebuildVexNums, markValidFinerVexNum.Array().ptr());

        int* RebuildVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVexAddress), sizeof(int) * RebuildEdgeArraySizeHost, stream));
        CHECKCUDA(hipMemsetAsync(RebuildVexAddress, 0, sizeof(int) * RebuildEdgeArraySizeHost, stream));

        void* d_temp_storage_3 = NULL;
        size_t temp_storage_bytes_3 = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_3, temp_storage_bytes_3, RebuildVexNums, RebuildVexAddress, RebuildEdgeArraySizeHost, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage_3, temp_storage_bytes_3, stream));
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_3, temp_storage_bytes_3, RebuildVexNums, RebuildVexAddress, RebuildEdgeArraySizeHost, stream);
        CHECKCUDA(hipFreeAsync(d_temp_storage_3, stream));

        CHECKCUDA(hipStreamSynchronize(stream));   // ��Ҫͬ��
        int RebuildLastVexAddr = -1;
        int RebuildLastVexNums = -1;
        CHECKCUDA(hipMemcpyAsync(&RebuildLastVexAddr, RebuildVexAddress + RebuildEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&RebuildLastVexNums, RebuildVexNums + RebuildEdgeArraySizeHost - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        int RebuildAllVexNums = RebuildLastVexAddr + RebuildLastVexNums;

        CHECKCUDA(hipStreamSynchronize(stream));   // ��Ҫͬ��
        //printf("depth = %d   RebuildAllVexNums = %d\n", i, RebuildAllVexNums);

        int* RebuildTriNums = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriNums), sizeof(int) * rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildTriNums, 0, sizeof(int) * rebuildDLevelCount, stream));

        int* RebuildCubeCatagory = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildCubeCatagory), sizeof(int) * rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildCubeCatagory, 0, sizeof(int) * rebuildDLevelCount, stream));

        dim3 block_9(128);
        dim3 grid_9(divUp(rebuildDLevelCount, block_9.x));
        device::generateTriNums << <grid_9, block_9, 0, stream >> > (RebuildArray, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildVvalue, RebuildTriNums, RebuildCubeCatagory);

        int* RebuildTriAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriAddress), sizeof(int)* rebuildDLevelCount, stream));
        CHECKCUDA(hipMemsetAsync(RebuildTriAddress, 0, sizeof(int)* rebuildDLevelCount, stream));

        void* d_temp_storage_4 = NULL;
        size_t temp_storage_bytes_4 = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_4, temp_storage_bytes_4, RebuildTriNums, RebuildTriAddress, rebuildDLevelCount, stream);
        CHECKCUDA(hipMallocAsync(&d_temp_storage_4, temp_storage_bytes_4, stream));
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage_4, temp_storage_bytes_4, RebuildTriNums, RebuildTriAddress, rebuildDLevelCount, stream);
        CHECKCUDA(hipFreeAsync(d_temp_storage_4, stream));

        Point3D<float>* RebuildVertexBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildVertexBuffer), sizeof(Point3D<float>)* RebuildAllVexNums, stream));
        //std::vector<Point3D<float>> RebuildVertexBufferHost;
        //RebuildVertexBufferHost.resize(RebuildAllVexNums);

        EdgeNode* RebuildValidEdgeArray = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidEdgeArray), sizeof(EdgeNode)* RebuildAllVexNums, stream));

        int* RebuildValidEdgeArraySize = NULL;
        int RebuildValidEdgeArraySizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidEdgeArraySize), sizeof(int), stream));

        void* d_temp_storage_5 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_5 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_5, temp_storage_bytes_5, RebuildEdgeArray, markValidFinerVexNum.Array().ptr(), RebuildValidEdgeArray, RebuildValidEdgeArraySize, RebuildEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_5, temp_storage_bytes_5, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_5, temp_storage_bytes_5, RebuildEdgeArray, markValidFinerVexNum.Array().ptr(), RebuildValidEdgeArray, RebuildValidEdgeArraySize, RebuildEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildValidEdgeArraySizeHost, RebuildValidEdgeArraySize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(d_temp_storage_5, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidEdgeArraySize, stream));

        int* RebuildValidVexAddress = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidVexAddress), sizeof(int)* RebuildAllVexNums, stream));

        int* RebuildValidVexAddressSize = NULL;
        int RebuildValidVexAddressSizeHost = -1;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildValidVexAddressSize), sizeof(int), stream));

        void* d_temp_storage_6 = NULL;    // �м���������꼴���ͷ�
        size_t temp_storage_bytes_6 = 0;  // �м����
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_6, temp_storage_bytes_6, RebuildVexAddress, markValidFinerVexNum.Array().ptr(), RebuildValidVexAddress, RebuildValidVexAddressSize, RebuildEdgeArraySizeHost, stream, false));	// ȷ����ʱ�豸�洢����
        CHECKCUDA(hipMallocAsync(&d_temp_storage_6, temp_storage_bytes_6, stream));
        CHECKCUDA(hipcub::DeviceSelect::Flagged(d_temp_storage_6, temp_storage_bytes_6, RebuildVexAddress, markValidFinerVexNum.Array().ptr(), RebuildValidVexAddress, RebuildValidVexAddressSize, RebuildEdgeArraySizeHost, stream, false));	// ɸѡ	
        CHECKCUDA(hipMemcpyAsync(&RebuildValidVexAddressSizeHost, RebuildValidVexAddressSize, sizeof(int), hipMemcpyDeviceToHost, stream));

        CHECKCUDA(hipFreeAsync(d_temp_storage_6, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidVexAddressSize, stream));

        dim3 block_10(128);
        dim3 grid_10(divUp(RebuildAllVexNums, block_10.x));
        device::generateSubdivideIntersectionPoint << <grid_10, block_10, 0, stream >> > (RebuildValidEdgeArray, RebuildVertexArray, RebuildArray, RebuildValidVexAddress, RebuildVvalue, RebuildAllVexNums, NodeArraySize, RebuildVertexBuffer);

        CHECKCUDA(hipFreeAsync(RebuildValidEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildValidVexAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildVvalue, stream));

        int RebuildLastTriAddr = -1;
        int RebuildLastTriNums = -1;

        CHECKCUDA(hipMemcpyAsync(&RebuildLastTriAddr, RebuildTriAddress + rebuildDLevelCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECKCUDA(hipMemcpyAsync(&RebuildLastTriNums, RebuildTriNums + rebuildDLevelCount - 1, sizeof(int), hipMemcpyDeviceToHost, stream));
    
        CHECKCUDA(hipStreamSynchronize(stream));
        int RebuildAllTriNums = RebuildLastTriAddr + RebuildLastTriNums;

        CHECKCUDA(hipFreeAsync(RebuildVexNums, stream));

        //printf("RebuildAllTriNums = %d\n", RebuildAllTriNums);

        TriangleIndex* RebuildTriangleBuffer = NULL;
        CHECKCUDA(hipMallocAsync(reinterpret_cast<void**>(&RebuildTriangleBuffer), sizeof(TriangleIndex) * RebuildAllTriNums, stream));
        //std::vector<int> RebuildTriangleBufferHost;
        //RebuildTriangleBufferHost.resize(RebuildAllTriNums * 3);

        CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��
        //printf("###############################   Depth = %d   #################################\n", i);
        dim3 block_11(128);
        dim3 grid_11(divUp(rebuildDLevelCount, block_11.x));
        device::generateSubdivideTrianglePos << <grid_11, block_11, 0, stream >> > (RebuildArray, depthNodeAddress[Constants::maxDepth_Host], rebuildDLevelCount, RebuildTriNums, RebuildCubeCatagory, RebuildVexAddress, RebuildTriAddress, RebuildTriangleBuffer);
        insertTriangle(RebuildVertexBuffer, RebuildAllVexNums, RebuildTriangleBuffer, RebuildAllTriNums, stream);

        //CHECKCUDA(hipMemcpyAsync(RebuildVertexBufferHost.data(), RebuildVertexBuffer, sizeof(Point3D<float>) * RebuildAllVexNums, hipMemcpyDeviceToHost, stream));
        //CHECKCUDA(hipMemcpyAsync(RebuildTriangleBufferHost.data(), RebuildTriangleBuffer, sizeof(int) * 3 * RebuildAllTriNums, hipMemcpyDeviceToHost, stream));

        //CHECKCUDA(hipStreamSynchronize(stream));   // ��ͬ��

        //insertTriangle(RebuildVertexBufferHost.data(), RebuildAllVexNums, RebuildTriangleBufferHost.data(), RebuildAllTriNums, mesh);

        CHECKCUDA(hipFreeAsync(fixedDepthNums, stream));
        CHECKCUDA(hipFreeAsync(depthNodeAddress_Device, stream));
        CHECKCUDA(hipFreeAsync(fixedDepthAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildDepthBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildCenterBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildVertexArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildEdgeArray, stream));
        CHECKCUDA(hipFreeAsync(RebuildVexAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriNums, stream));
        CHECKCUDA(hipFreeAsync(RebuildCubeCatagory, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriAddress, stream));
        CHECKCUDA(hipFreeAsync(RebuildVertexBuffer, stream));
        CHECKCUDA(hipFreeAsync(RebuildTriangleBuffer, stream));
    }
}
