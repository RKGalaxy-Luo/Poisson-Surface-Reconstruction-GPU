/*****************************************************************//**
 * \file   CommonUtils.cu
 * \brief  �ļ���Ҫ����һЩ���õ�cuda���ߺ���
 * 
 * \author LUO
 * \date   January 12th 2024
 *********************************************************************/
#include "CommonUtils.h"


hipCtx_t SparseSurfelFusion::initCudaContext(int selected_device) {
    //��ʼ��Cuda������API
    CHECKCUDADRIVER(hipInit(0));

    //Query the device
    int device_count = 0;
    CHECKCUDADRIVER(hipGetDeviceCount(&device_count));
    for (auto dev_idx = 0; dev_idx < device_count; dev_idx++) {
        char dev_name[256] = { 0 };
        CHECKCUDADRIVER(hipDeviceGetName(dev_name, 256, dev_idx));
        printf("device %d: %s\n", dev_idx, dev_name);
    }

    //ѡ��GPU
    printf("�豸 %d ���������д�����.\n", selected_device);
    hipDevice_t cuda_device;
    CHECKCUDADRIVER(hipDeviceGet(&cuda_device, selected_device));

    //����cuda������
    hipCtx_t cuda_context;
    CHECKCUDADRIVER(hipCtxCreate(&cuda_context, hipDeviceScheduleAuto, cuda_device));
    return cuda_context;
}

void SparseSurfelFusion::destroyCudaContext(hipCtx_t context)
{
    hipDeviceSynchronize();
    CHECKCUDADRIVER(hipCtxDestroy(context));
}



void SparseSurfelFusion::createDefault2DTextureDescriptor(hipTextureDesc& descriptor)
{
    memset(&descriptor, 0, sizeof(descriptor));
    // ����Ѱַģʽ��ʹ��3��ά��(ʵ����ֻʹ��2ά)
    descriptor.addressMode[0] = hipAddressModeBorder;  // �ڱ߽�֮�ⷵ��0
    descriptor.addressMode[1] = hipAddressModeBorder;
    descriptor.addressMode[2] = hipAddressModeBorder;
    // �������ȡʱҪʹ�õĹ���ģʽ
    descriptor.filterMode = hipFilterModePoint;        // ���ڽ���ֵ--hipFilterModePoint       ˫���Բ�ֵ--hipFilterModeLinear
    // ָ���Ƿ�Ӧ����������ת��Ϊ������
    descriptor.readMode = hipReadModeElementType;      // ��������ָ�����������Ͷ�����ȫ��ת����float
    // �Ƿ����������׼��
    descriptor.normalizedCoords = 0;                    // ��ʹ�ù�һ�������ڴ�

}

void SparseSurfelFusion::createDefault2DResourceDescriptor(hipResourceDesc& descriptor, hipArray_t& hipArray)
{
    memset(&descriptor, 0, sizeof(hipResourceDesc));   // ��Դ�����ӳ�ֵΪ0
    // ʹ��CUDA����--hipResourceTypeArray      
    // ʹ��CUDAӳ������--hipResourceTypeMipmappedArray      
    // ʹ���豸��һ�������ڴ�--hipResourceTypeLinear
    // ʹ���豸��һ��2D����Դ
    descriptor.resType = hipResourceTypeArray;         
    descriptor.res.array.array = hipArray;             // ��ֵ���ڴ�θ���
}

void SparseSurfelFusion::createDepthTexture(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipArray_t& hipArray)
{
    // ������������
    hipTextureDesc depth_texture_desc;
    createDefault2DTextureDescriptor(depth_texture_desc);
    // ����ͨ������(ֻ��һ��ͨ�������ݣ�����������uint16)
    hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned); // 16bit���޷�������
    // ����cuda����
    CHECKCUDA(hipMallocArray(&hipArray, &depth_channel_desc, cols, rows));
    // ������Դ����
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ
    // ���������ڴ�
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
}

void SparseSurfelFusion::createDepthTextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //��������
    hipTextureDesc depth_texture_description;
    createDefault2DTextureDescriptor(depth_texture_description);
    //����ͨ������
    hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
    //����cuda����
    CHECKCUDA(hipMallocArray(&hipArray, &depth_channel_desc, cols, rows));
    //������Դdesc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ
    //��������
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_description, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createDepthTextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& collect)
{
    createDepthTextureSurface(rows, cols,collect.texture, collect.surface, collect.hipArray);
}

void SparseSurfelFusion::createFloat1TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //��������
    hipTextureDesc float1_texture_desc;
    createDefault2DTextureDescriptor(float1_texture_desc);
    //����ͨ��������ʹ��ָ�����ͷ���ͨ�������ӣ�������ÿһ��ͨ��������bit�� (����Ϊ1��ͨ������ͨ������λ��Ϊ32bit)
    hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    //����cuda����
    CHECKCUDA(hipMallocArray(&hipArray, &float1_channel_desc, cols, rows));
    //������Դdesc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ
    //��������
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat1TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat1TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::createFloat2TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //��������
    hipTextureDesc float2_texture_desc;
    createDefault2DTextureDescriptor(float2_texture_desc);
    //����ͨ��������ʹ��ָ�����ͷ���ͨ�������ӣ�������ÿһ��ͨ��������bit�� (����Ϊ2��ͨ������ͨ������λ��Ϊ32bit)
    hipChannelFormatDesc float2_channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
    //����cuda����
    CHECKCUDA(hipMallocArray(&hipArray, &float2_channel_desc, cols, rows));
    //������Դdesc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ
    //��������
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float2_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat2TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat2TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::createUChar1TextureSurface(const unsigned rows, const unsigned cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    //��������
    hipTextureDesc uchar1_texture_desc;
    createDefault2DTextureDescriptor(uchar1_texture_desc);
    //����ͨ��������ʹ��ָ�����ͷ���ͨ�������ӣ�������ÿһ��ͨ��������bit�� (����Ϊ1��ͨ������ͨ������λ��Ϊ8bit)
    hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    //����cuda����
    CHECKCUDA(hipMallocArray(&hipArray, &uchar1_channel_desc, cols, rows));
    //������Դdesc
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ
    //��������
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createUChar1TextureSurface(const unsigned rows, const unsigned cols, CudaTextureSurface& textureCollect)
{
    createUChar1TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}


void SparseSurfelFusion::createFloat4TextureSurface(const unsigned int rows, const unsigned int cols, hipTextureObject_t& texture, hipSurfaceObject_t& surface, hipArray_t& hipArray)
{
    // ��������ʼ������������
    hipTextureDesc float4_texture_desc; // ����������
    createDefault2DTextureDescriptor(float4_texture_desc); // �����������ӳ�ʼ��

    // ����ͨ��������ʹ��ָ�����ͷ���ͨ�������ӣ�������ÿһ��ͨ��������bit��
    // ʹ��float���ͷ��������ӣ�����ÿ��ͨ����������32bitλ
    hipChannelFormatDesc float4_channel_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); // ������һ��float���͵�ͨ��������

    // ����ͨ��������float4_channel_desc����cuda���ڴ�
    CHECKCUDA(hipMallocArray(&hipArray, &float4_channel_desc, cols, rows));

    // ������Դ������
    hipResourceDesc resource_desc;
    createDefault2DResourceDescriptor(resource_desc, hipArray); // ��ʼ����Դ�����Ӳ�����Դ����hipArray��ֵ��ȥ

    // ���������ڴ�
    CHECKCUDA(hipCreateTextureObject(&texture, &resource_desc, &float4_texture_desc, 0));
    // ��������ڴ�
    CHECKCUDA(hipCreateSurfaceObject(&surface, &resource_desc));
}

void SparseSurfelFusion::createFloat4TextureSurface(const unsigned int rows, const unsigned int cols, CudaTextureSurface& textureCollect)
{
    createFloat4TextureSurface(rows, cols, textureCollect.texture, textureCollect.surface, textureCollect.hipArray);
}

void SparseSurfelFusion::releaseTextureCollect(CudaTextureSurface& textureCollect)
{
    CHECKCUDA(hipDestroyTextureObject(textureCollect.texture));
    CHECKCUDA(hipDestroySurfaceObject(textureCollect.surface));
    CHECKCUDA(hipFreeArray(textureCollect.hipArray));
}

void SparseSurfelFusion::query2DTextureExtent(hipTextureObject_t texture, unsigned int& width, unsigned int& height)
{
    hipResourceDesc texture_res;
    cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
    hipArray_t cu_array = texture_res.res.array.array;
    hipChannelFormatDesc channel_desc;
    hipExtent extent;
    unsigned int flag;
    cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));
    width = extent.width;
    height = extent.height;
}
